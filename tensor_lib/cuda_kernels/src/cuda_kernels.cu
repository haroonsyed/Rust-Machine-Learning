#include "hip/hip_runtime.h"
#include <chrono>

#include "cuda_kernels.cuh"

/////////////////////
/// TEST FUNCTIONS
/////////////////////
__global__ void test_kernel() {
    printf("Hello from the kernel!\n");
}

__global__ void test_kernel_2(int* result) {
    *result = 8;
}

void test() {
    int result;
    int* d_result;
    test_kernel<<<1, 1>>>();
    gpuErrchk(hipPeekAtLastError());
    hipDeviceSynchronize();
    if (hipPeekAtLastError() != hipSuccess) {
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
    }

    hipError_t err = hipMalloc((void**)&d_result, sizeof(int));
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }

    test_kernel_2<<<1, 1>>>(d_result);
    if (hipPeekAtLastError() != hipSuccess) {
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
    }

    err = hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    hipFree(d_result);

    printf("Result: %d\n", result);
    printf("Finished Running Kernels.");
}

void test_array_fill(float* buffer, size_t length) {
    for (size_t i = 0; i < length; i++) {
        buffer[i] = i;
    }
}

void cuda_synchronize() {
    hipDeviceSynchronize();
}

//////////////////////////
/// Device Functions
//////////////////////////
__device__ float atomicMultiply(float* address, float val) {
    // We will need to use atomicCAS, since there is not a built in
    float expected_old = *address;
    float actual_old = __int_as_float(atomicCAS((int*)address, __float_as_int(expected_old), __float_as_int(expected_old * val)));
    while (expected_old != actual_old && !__isnanf(expected_old)) {
        expected_old = actual_old;
        actual_old = __int_as_float(atomicCAS((int*)address, __float_as_int(expected_old), __float_as_int(expected_old * val)));
    }
}

__device__ float atomicDivide(float* address, float val) {
    // We will need to use atomicCAS, since there is not a built in
    float expected_old = *address;
    float actual_old = __int_as_float(atomicCAS((int*)address, __float_as_int(expected_old), __float_as_int(expected_old / val)));
    while (expected_old != actual_old && !__isnanf(expected_old)) {
        expected_old = actual_old;
        actual_old = __int_as_float(atomicCAS((int*)address, __float_as_int(expected_old), __float_as_int(expected_old / val)));
    }
}

//////////////////////////
/// Matrix Operations API
//////////////////////////
__global__ void element_add_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;
    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] + mat2[i][j]
        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] + mat2_buffer[index];
    }
}
size_t cuda_element_add(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Create output buffer`
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);
    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_mat2_buffer = get_matrix_gpu_address(mat2_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);
    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);
    // Run the kernels
    element_add_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());
    // Return result matrix id
    return out_mat_id;
}
// Each block handles one matrix
__global__ void cuda_element_add_packed_kernel(float** mat1_buffers, float** mat2_buffers, float** out_buffers, int mat_rows, int mat_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat1_buffer = mat1_buffers[current_matrix];
    const float* mat2_buffer = mat2_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;
            out_buffer[index] = mat1_buffer[index] + mat2_buffer[index];
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}
// Each block handles one matrix
__global__ void cuda_element_add_packed_inplace_kernel(float** mat1_buffers, float** mat2_buffers, int mat_rows, int mat_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    float* mat1_buffer = mat1_buffers[current_matrix];
    const float* mat2_buffer = mat2_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;

            // Atomic because mat1 may be used multiple times
            atomicAdd(&mat1_buffer[index], mat2_buffer[index]);
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}
void cuda_element_add_packed(size_t* mat1_ids, size_t* mat2_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Get the gpu buffers to operate on
    float** pinned_mat1_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_mat2_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat1_buffers_ptr[i] = get_matrix_gpu_address(mat1_ids[i]);
        pinned_mat2_buffers_ptr[i] = get_matrix_gpu_address(mat2_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(3);
    float** gpu_mat1_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_mat2_buffers_dp = (float**)kernel_arg_device_pointers[1];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[2];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat1_buffers_dp, pinned_mat1_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_mat2_buffers_dp, pinned_mat2_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_element_add_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffers_dp, gpu_mat2_buffers_dp, gpu_out_buffers_dp, mat_rows, mat_cols);
    gpuErrchk(hipPeekAtLastError());
}

void cuda_element_add_packed_inplace(size_t* mat1_ids, size_t* mat2_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    // Get the gpu buffers to operate on
    float** pinned_mat1_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_mat2_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat1_buffers_ptr[i] = get_matrix_gpu_address(mat1_ids[i]);
        pinned_mat2_buffers_ptr[i] = get_matrix_gpu_address(mat2_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(2);
    float** gpu_mat1_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_mat2_buffers_dp = (float**)kernel_arg_device_pointers[1];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat1_buffers_dp, pinned_mat1_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_mat2_buffers_dp, pinned_mat2_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_element_add_packed_inplace_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffers_dp, gpu_mat2_buffers_dp, mat_rows, mat_cols);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void element_subtract_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;
    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] - mat2[i][j]

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] - mat2_buffer[index];
    }
}

size_t cuda_element_subtract(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_mat2_buffer = get_matrix_gpu_address(mat2_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    element_subtract_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Each block handles one matrix
__global__ void cuda_element_subtract_packed_kernel(float** mat1_buffers, float** mat2_buffers, float** out_buffers, int mat_rows, int mat_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat1_buffer = mat1_buffers[current_matrix];
    const float* mat2_buffer = mat2_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;
            out_buffer[index] = mat1_buffer[index] - mat2_buffer[index];
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}
// Each block handles one matrix
__global__ void cuda_element_subtract_packed_inplace_kernel(float** mat1_buffers, float** mat2_buffers, int mat_rows, int mat_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    float* mat1_buffer = mat1_buffers[current_matrix];
    const float* mat2_buffer = mat2_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;

            // Atomic because mat1 may be used multiple times
            atomicAdd(&mat1_buffer[index], -mat2_buffer[index]);
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}
void cuda_element_subtract_packed(size_t* mat1_ids, size_t* mat2_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Get the gpu buffers to operate on
    float** pinned_mat1_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_mat2_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat1_buffers_ptr[i] = get_matrix_gpu_address(mat1_ids[i]);
        pinned_mat2_buffers_ptr[i] = get_matrix_gpu_address(mat2_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(3);
    float** gpu_mat1_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_mat2_buffers_dp = (float**)kernel_arg_device_pointers[1];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[2];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat1_buffers_dp, pinned_mat1_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_mat2_buffers_dp, pinned_mat2_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_element_subtract_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffers_dp, gpu_mat2_buffers_dp, gpu_out_buffers_dp, mat_rows, mat_cols);
    gpuErrchk(hipPeekAtLastError());
}

void cuda_element_subtract_packed_inplace(size_t* mat1_ids, size_t* mat2_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    // Get the gpu buffers to operate on
    float** pinned_mat1_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_mat2_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat1_buffers_ptr[i] = get_matrix_gpu_address(mat1_ids[i]);
        pinned_mat2_buffers_ptr[i] = get_matrix_gpu_address(mat2_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(3);
    float** gpu_mat1_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_mat2_buffers_dp = (float**)kernel_arg_device_pointers[1];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat1_buffers_dp, pinned_mat1_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_mat2_buffers_dp, pinned_mat2_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_element_subtract_packed_inplace_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffers_dp, gpu_mat2_buffers_dp, mat_rows, mat_cols);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void element_multiply_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] * mat2[i][j]

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] * mat2_buffer[index];
    }
}

size_t cuda_element_multiply(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_mat2_buffer = get_matrix_gpu_address(mat2_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    element_multiply_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Each block handles one matrix
__global__ void cuda_element_multiply_packed_kernel(float** mat1_buffers, float** mat2_buffers, float** out_buffers, int mat_rows, int mat_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat1_buffer = mat1_buffers[current_matrix];
    const float* mat2_buffer = mat2_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;
            out_buffer[index] = mat1_buffer[index] * mat2_buffer[index];
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}
// Each block handles one matrix
__global__ void cuda_element_multiply_packed_inplace_kernel(float** mat1_buffers, float** mat2_buffers, int mat_rows, int mat_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    float* mat1_buffer = mat1_buffers[current_matrix];
    const float* mat2_buffer = mat2_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;

            // Atomic because mat1 may be used multiple times
            atomicMultiply(&mat1_buffer[index], mat2_buffer[index]);
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}
void cuda_element_multiply_packed(size_t* mat1_ids, size_t* mat2_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);
    // Get the gpu buffers to operate on
    float** pinned_mat1_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_mat2_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat1_buffers_ptr[i] = get_matrix_gpu_address(mat1_ids[i]);
        pinned_mat2_buffers_ptr[i] = get_matrix_gpu_address(mat2_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(3);
    float** gpu_mat1_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_mat2_buffers_dp = (float**)kernel_arg_device_pointers[1];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[2];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat1_buffers_dp, pinned_mat1_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_mat2_buffers_dp, pinned_mat2_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_element_multiply_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffers_dp, gpu_mat2_buffers_dp, gpu_out_buffers_dp, mat_rows, mat_cols);
    gpuErrchk(hipPeekAtLastError());
}

void cuda_element_multiply_packed_inplace(size_t* mat1_ids, size_t* mat2_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    // Get the gpu buffers to operate on
    float** pinned_mat1_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_mat2_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat1_buffers_ptr[i] = get_matrix_gpu_address(mat1_ids[i]);
        pinned_mat2_buffers_ptr[i] = get_matrix_gpu_address(mat2_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(3);
    float** gpu_mat1_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_mat2_buffers_dp = (float**)kernel_arg_device_pointers[1];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat1_buffers_dp, pinned_mat1_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_mat2_buffers_dp, pinned_mat2_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_element_multiply_packed_inplace_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffers_dp, gpu_mat2_buffers_dp, mat_rows, mat_cols);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void element_divide_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] * mat2[i][j]

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] / mat2_buffer[index];
    }
}

size_t cuda_element_divide(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_mat2_buffer = get_matrix_gpu_address(mat2_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    element_divide_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Each block handles one matrix
__global__ void cuda_element_divide_packed_kernel(float** mat1_buffers, float** mat2_buffers, float** out_buffers, int mat_rows, int mat_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat1_buffer = mat1_buffers[current_matrix];
    const float* mat2_buffer = mat2_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;
            out_buffer[index] = mat1_buffer[index] / mat2_buffer[index];
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}
// Each block handles one matrix
__global__ void cuda_element_divide_packed_inplace_kernel(float** mat1_buffers, float** mat2_buffers, int mat_rows, int mat_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    float* mat1_buffer = mat1_buffers[current_matrix];
    const float* mat2_buffer = mat2_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;

            // Atomic because mat1 may be used multiple times
            atomicDivide(&mat1_buffer[index], mat2_buffer[index]);
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}
void cuda_element_divide_packed(size_t* mat1_ids, size_t* mat2_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Get the gpu buffers to operate on
    float** pinned_mat1_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_mat2_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat1_buffers_ptr[i] = get_matrix_gpu_address(mat1_ids[i]);
        pinned_mat2_buffers_ptr[i] = get_matrix_gpu_address(mat2_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(3);
    float** gpu_mat1_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_mat2_buffers_dp = (float**)kernel_arg_device_pointers[1];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[2];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat1_buffers_dp, pinned_mat1_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_mat2_buffers_dp, pinned_mat2_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_element_divide_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffers_dp, gpu_mat2_buffers_dp, gpu_out_buffers_dp, mat_rows, mat_cols);
    gpuErrchk(hipPeekAtLastError());
}

void cuda_element_divide_packed_inplace(size_t* mat1_ids, size_t* mat2_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    // Get the gpu buffers to operate on
    float** pinned_mat1_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_mat2_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat1_buffers_ptr[i] = get_matrix_gpu_address(mat1_ids[i]);
        pinned_mat2_buffers_ptr[i] = get_matrix_gpu_address(mat2_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(2);
    float** gpu_mat1_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_mat2_buffers_dp = (float**)kernel_arg_device_pointers[1];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat1_buffers_dp, pinned_mat1_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_mat2_buffers_dp, pinned_mat2_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_element_divide_packed_inplace_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffers_dp, gpu_mat2_buffers_dp, mat_rows, mat_cols);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void scalar_multiply_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float scalar, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] * scalar

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] * scalar;
    }
}

size_t cuda_scalar_multiply(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, float scalar, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    scalar_multiply_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, scalar, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Each block handles one matrix
__global__ void cuda_scalar_multiply_packed_kernel(float** mat_buffers, float** out_buffers, int mat_rows, int mat_cols, float scalar) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;
            out_buffer[index] = mat_buffer[index] * scalar;
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}
// Each block handles one matrix
__global__ void cuda_scalar_multiply_packed_inplace_kernel(float** mat_buffers, int mat_rows, int mat_cols, float scalar) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    float* mat_buffer = mat_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;

            // Atomic because mat1 may be used multiple times
            atomicMultiply(&mat_buffer[index], scalar);
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}
void cuda_scalar_multiply_packed(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, float scalar) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Grab device pointers and put in pinned memory to upload as device array
    float** pinned_mat1_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat1_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(2);
    float** gpu_mat1_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[1];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat1_buffers_dp, pinned_mat1_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_scalar_multiply_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffers_dp, gpu_out_buffers_dp, mat_rows, mat_cols, scalar);
    gpuErrchk(hipPeekAtLastError());
}

void cuda_scalar_multiply_packed_inplace(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, float scalar) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    // Grab device pointers and put in pinned memory to upload as device array
    float** pinned_mat1_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat1_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(1);
    float** gpu_mat1_buffers_dp = (float**)kernel_arg_device_pointers[0];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat1_buffers_dp, pinned_mat1_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_scalar_multiply_packed_inplace_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffers_dp, mat_rows, mat_cols, scalar);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void scalar_divide_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float scalar, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] * scalar

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] / scalar;
    }
}

size_t cuda_scalar_divide(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, float scalar, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    scalar_divide_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, scalar, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Each block handles one matrix
__global__ void cuda_scalar_divide_packed_kernel(float** mat_buffers, float** out_buffers, int mat_rows, int mat_cols, float scalar) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;
            out_buffer[index] = mat_buffer[index] / scalar;
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}
// Each block handles one matrix
__global__ void cuda_scalar_divide_packed_inplace_kernel(float** mat_buffers, int mat_rows, int mat_cols, float scalar) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    float* mat_buffer = mat_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;

            // Atomic because mat1 may be used multiple times
            atomicDivide(&mat_buffer[index], scalar);
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}
void cuda_scalar_divide_packed(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, float scalar) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Grab device pointers and put in pinned memory to upload as device array
    float** pinned_mat1_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat1_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(2);
    float** gpu_mat1_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[1];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat1_buffers_dp, pinned_mat1_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_scalar_divide_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffers_dp, gpu_out_buffers_dp, mat_rows, mat_cols, scalar);
    gpuErrchk(hipPeekAtLastError());
}

void cuda_scalar_divide_packed_inplace(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, float scalar) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    // Grab device pointers and put in pinned memory to upload as device array
    float** pinned_mat1_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat1_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(1);
    float** gpu_mat1_buffers_dp = (float**)kernel_arg_device_pointers[0];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat1_buffers_dp, pinned_mat1_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_scalar_divide_packed_inplace_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffers_dp, mat_rows, mat_cols, scalar);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void scalar_add_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float scalar, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] * scalar

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] + scalar;
    }
}

size_t cuda_scalar_add(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, float scalar, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    scalar_add_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, scalar, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Each block handles one matrix
__global__ void cuda_scalar_add_packed_kernel(float** mat_buffers, float** out_buffers, int mat_rows, int mat_cols, float scalar) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;
            out_buffer[index] = mat_buffer[index] + scalar;
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}
// Each block handles one matrix
__global__ void cuda_scalar_add_packed_inplace_kernel(float** mat_buffers, int mat_rows, int mat_cols, float scalar) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    float* mat_buffer = mat_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;

            // Atomic because mat1 may be used multiple times
            atomicAdd(&mat_buffer[index], scalar);
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}
void cuda_scalar_add_packed(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, float scalar) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Grab device pointers and put in pinned memory to upload as device array
    float** pinned_mat1_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat1_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(2);
    float** gpu_mat1_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[1];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat1_buffers_dp, pinned_mat1_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_scalar_add_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffers_dp, gpu_out_buffers_dp, mat_rows, mat_cols, scalar);
    gpuErrchk(hipPeekAtLastError());
}

void cuda_scalar_add_packed_inplace(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, float scalar) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    // Grab device pointers and put in pinned memory to upload as device array
    float** pinned_mat1_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat1_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(1);
    float** gpu_mat1_buffers_dp = (float**)kernel_arg_device_pointers[0];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat1_buffers_dp, pinned_mat1_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_scalar_add_packed_inplace_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffers_dp, mat_rows, mat_cols, scalar);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void scalar_subtract_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float scalar, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] * scalar

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] - scalar;
    }
}

size_t cuda_scalar_subtract(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, float scalar, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    scalar_subtract_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, scalar, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Each block handles one matrix
__global__ void cuda_scalar_subtract_packed_kernel(float** mat_buffers, float** out_buffers, int mat_rows, int mat_cols, float scalar) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;
            out_buffer[index] = mat_buffer[index] - scalar;
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}
// Each block handles one matrix
__global__ void cuda_scalar_subtract_packed_inplace_kernel(float** mat_buffers, int mat_rows, int mat_cols, float scalar) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    float* mat_buffer = mat_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;

            // Atomic because mat1 may be used multiple times
            atomicAdd(&mat_buffer[index], -scalar);
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}
void cuda_scalar_subtract_packed(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, float scalar) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Grab device pointers and put in pinned memory to upload as device array
    float** pinned_mat1_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat1_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(2);
    float** gpu_mat1_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[1];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat1_buffers_dp, pinned_mat1_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_scalar_subtract_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffers_dp, gpu_out_buffers_dp, mat_rows, mat_cols, scalar);
    gpuErrchk(hipPeekAtLastError());
}

void cuda_scalar_subtract_packed_inplace(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, float scalar) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    // Grab device pointers and put in pinned memory to upload as device array
    float** pinned_mat1_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat1_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(1);
    float** gpu_mat1_buffers_dp = (float**)kernel_arg_device_pointers[0];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat1_buffers_dp, pinned_mat1_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_scalar_subtract_packed_inplace_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffers_dp, mat_rows, mat_cols, scalar);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void matrix_multiply_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][:] weighted sum mat2[:][j]
        // Where common dimension : is mat1col/mat2row

        float weighted_sum = 0.0;
        for (int common = 0; common < mat1_cols; common++) {
            // mat1[i][common]
            int mat1_index = mat1_cols * tidX + common;
            // mat1[common][j]
            int mat2_index = mat2_cols * common + tidY;

            weighted_sum += mat1_buffer[mat1_index] * mat2_buffer[mat2_index];
        }

        int output_index = tidX * out_cols + tidY;
        out_buffer[output_index] = weighted_sum;
    }
}

__global__ void matrix_multiply_kernel_2(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    // Go by col row instead of row col. Enabled memory coalescing
    const int col = blockDim.x * blockIdx.x + threadIdx.x;
    const int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (row >= out_rows || col >= out_cols) {
        return;
    }

    // O[i][j] = mat1[i][:] weighted sum mat2[:][j]
    // Where common dimension : is mat1col/mat2row

    float weighted_sum = 0.0;
    for (int common = 0; common < mat1_cols; common++) {
        // mat1[i][common]
        int mat1_index = mat1_cols * row + common;
        // mat1[common][j]
        int mat2_index = mat2_cols * common + col;

        weighted_sum += mat1_buffer[mat1_index] * mat2_buffer[mat2_index];
    }

    const int output_index = row * out_cols + col;
    out_buffer[output_index] = weighted_sum;
}

__global__ void matrix_multiply_kernel_3(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    const int block_dim = 32;
    const int block_area = block_dim * block_dim;

    // Block tiling with shared memory
    __shared__ float s_mat1[block_area];
    __shared__ float s_mat2[block_area];

    const int block_row = blockIdx.y;
    const int block_col = blockIdx.x;

    int mat1_block_pos = block_row * block_dim * mat1_cols;
    int mat2_block_pos = block_col * block_dim;
    int out_block_pos = block_row * block_dim * out_cols + block_col * block_dim;

    // So within our block we are gonna figure out this thread's position
    int thread_row = threadIdx.y;
    int thread_col = threadIdx.x;

    int out_row = block_row * block_dim + thread_row;
    int out_col = block_col * block_dim + thread_col;
    if (out_row >= out_rows || out_col >= out_cols) {
        return;
    }

    float weighted_sum = 0.0;
    int common_partial_block = mat1_cols % block_dim;
    int common_in_block = mat1_cols - common_partial_block;
    for (int k = 0; k < common_in_block; k += block_dim) {
        s_mat1[thread_row * block_dim + thread_col] = mat1_buffer[mat1_block_pos + thread_row * mat1_cols + thread_col];
        s_mat2[thread_row * block_dim + thread_col] = mat2_buffer[mat2_block_pos + thread_row * mat2_cols + thread_col];
        __syncthreads();

        mat1_block_pos += block_dim;
        mat2_block_pos += block_dim * mat2_cols;
        for (int i = 0; i < block_dim; i++) {
            weighted_sum += s_mat1[thread_row * block_dim + i] * s_mat2[i * block_dim + thread_col];
        }
        __syncthreads();
    }

    // Handle partial block case
    s_mat1[thread_row * block_dim + thread_col] = mat1_buffer[mat1_block_pos + thread_row * mat1_cols + thread_col];
    s_mat2[thread_row * block_dim + thread_col] = mat2_buffer[mat2_block_pos + thread_row * mat2_cols + thread_col];
    __syncthreads();

    mat1_block_pos += block_dim;
    mat2_block_pos += block_dim * mat2_cols;
    for (int i = 0; i < common_partial_block; i++) {
        weighted_sum += s_mat1[thread_row * block_dim + i] * s_mat2[i * block_dim + thread_col];
    }

    out_buffer[out_block_pos + (thread_row * out_cols) + thread_col] = weighted_sum;
}

// block_M is rows in mat1 shared block
// block_N is cols in mat2 shared block
// block_k is shared dimensions for shared block. Also the # of results each thread will compute in C
// For this to work we want the shared dimension block_K to be smaller than block_M and block_N
// This way, multiple threads reuse sections from mat1 and mat2 ,with more output work
// Example: bK is 8 while bM and bN are 64. Output is a 64x64 area.
//          So you can spin up 512 threads per block. They load vram->shared
//          Then each thread can work on 8 pieces of the output 64x64 area (64*64/8 = 512)
template <const int block_M, const int block_N, const int block_K>
__global__ void matrix_multiply_kernel_4(int M, int N, int K, float* mat1_buffer, float* mat2_buffer, float* out_buffer) {
    // Block tiling with shared memory
    // Each one of these threads will handle #block_K output result columns
    __shared__ float s_mat1[block_M * block_K];
    __shared__ float s_mat2[block_K * block_N];

    float thread_results[block_K] = {0.0};

    const int block_row = blockIdx.y;
    const int block_col = blockIdx.x;

    // Get starting positions of each block
    int mat1_block_pos = block_row * block_M * K;
    int mat2_block_pos = block_col * block_N;
    int out_block_pos = block_row * block_M * N + block_col * block_N;

    // Used to track if out of bounds
    const int mat1_load_index_row = block_row * block_M + threadIdx.x;
    const int mat2_load_index_col = block_col * block_N + threadIdx.x;
    int mat_common_index = threadIdx.y;
    const bool exceeded_mat1_row = mat1_load_index_row >= M;
    const bool exceeded_mat2_col = mat2_load_index_col >= N;

    // outer loop over block tiles
    for (unsigned int common_block = 0; common_block < K; common_block += block_K) {
        const int within_mat1 = (int)!(exceeded_mat1_row || mat_common_index >= K);
        const int within_mat2 = (int)!(mat_common_index >= K || exceeded_mat2_col);
        int mat1_load_index = mat1_block_pos + threadIdx.x * K + threadIdx.y;
        int mat2_load_index = mat2_block_pos + threadIdx.y * N + threadIdx.x;

        // Prevent loading OOB
        mat1_load_index *= within_mat1;
        mat2_load_index *= within_mat2;

        // Load block data into shared memory. Load 0 is OOB.
        s_mat1[threadIdx.x * block_K + threadIdx.y] = mat1_buffer[mat1_load_index] * within_mat1;
        s_mat2[threadIdx.y * block_N + threadIdx.x] = mat2_buffer[mat2_load_index] * within_mat2;
        __syncthreads();

        // Advance block
        mat1_block_pos += block_K;
        mat2_block_pos += block_K * N;
        mat_common_index += block_K;

        // Go through common dimensions of block (across row of mat1 and down col of mat2)
        for (unsigned int block_common_index = 0; block_common_index < block_K; ++block_common_index) {
            const float shared_mat2_val = s_mat2[block_common_index * block_N + threadIdx.x];

            // Now this thread will accumulate the result for each t_row in the t_col of C
            for (unsigned int result_index = 0; result_index < block_K; ++result_index) {
                thread_results[result_index] +=
                    s_mat1[(threadIdx.y * block_K + result_index) * block_K + block_common_index] * shared_mat2_val;
            }
        }
        __syncthreads();
    }

    // Write results with bounds checking
    const int out_index_row = block_row * block_M + threadIdx.y * block_K;
    const int out_index_col = block_col * block_N + threadIdx.x;

    for (int i = 0; i < block_K; i++) {
        if (out_index_row + i < M && out_index_col < N) {
            out_buffer[out_block_pos + (threadIdx.y * block_K + i) * N + threadIdx.x] = thread_results[i];
        }
    }
}

// block_M is rows in mat1 shared block
// block_N is cols in mat2 shared block
// block_k is shared dimensions for shared block.
// The thread will calculate block_k * block_k results (So now a 2d version of v3)
// For this to work we want the shared dimension block_K to be extremely smaller than block_M and block_N
// This way, multiple threads reuse sections from mat1 and mat2 ,with more output work
// Example: bK is 8 while bM and bN are 128. Output is a 128x128 area.
//          So you can spin up 256 threads per block. They load vram->shared
//          Then each thread can work on 8x8 pieces of the output 128x128 area (128x128/64 = 256)
//          You might be wondering why not 512 threads like previously?
//          Well that increases the mem requirements per block, reducing occupancy.
template <const int block_M, const int block_N, const int block_K>
__global__ void matrix_multiply_kernel_5(int M, int N, int K, float* __restrict__ mat1_buffer, float* __restrict__ mat2_buffer, float* __restrict__ out_buffer) {
    // 2D Block tiling with shared memory
    __shared__ float s_mat1[block_M * block_K];
    __shared__ float s_mat2[block_K * block_N];

    float thread_results[block_K * block_K] = {0.0};

    const int block_row = blockIdx.y;
    const int block_col = blockIdx.x;

    // Output within block details
    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int out_block_row = tid / (block_M / block_K);
    const int out_block_col = tid % (block_N / block_K);

    const int num_threads_per_block = blockDim.x * blockDim.y;
    const int num_elements_to_load = (block_M * block_K) / num_threads_per_block;

    const int stride_mat1 = num_threads_per_block / block_K;
    const int stride_mat2 = num_threads_per_block / block_N;

    int mat1_pos = block_row * block_M * K;
    int mat2_pos = block_col * block_N;

// outer loop over block tiles
#pragma unroll
    for (int common_block = 0; common_block < K; common_block += block_K) {
#pragma unroll 4
        for (int i = 0; i < num_elements_to_load; i++) {
            const int mat1_row_within_block = (threadIdx.x + stride_mat1 * i);
            const int mat1_col_within_block = threadIdx.y;
            const int mat2_row_within_block = (threadIdx.y / num_elements_to_load) + i * stride_mat2;
            const int mat2_col_within_block = (threadIdx.y % num_elements_to_load) * blockDim.x + threadIdx.x;

            const int mat1_load_index_row = block_row * block_M + mat1_row_within_block;
            const int mat1_load_index_col = common_block + mat1_col_within_block;
            const int mat2_load_index_row = common_block + mat2_row_within_block;
            const int mat2_load_index_col = block_col * block_N + mat2_col_within_block;

            const bool exceeded_mat1_row = mat1_load_index_row >= M;
            const bool exceeded_mat1_col = mat1_load_index_col >= K;
            const bool exceeded_mat2_row = mat2_load_index_row >= K;
            const bool exceeded_mat2_col = mat2_load_index_col >= N;

            const int within_mat1 = (int)!(exceeded_mat1_row || exceeded_mat1_col);
            const int within_mat2 = (int)!(exceeded_mat2_row || exceeded_mat2_col);
            int mat1_load_index = mat1_pos + mat1_row_within_block * K + mat1_col_within_block;
            int mat2_load_index = mat2_pos + mat2_row_within_block * N + mat2_col_within_block;

            mat1_load_index *= within_mat1;
            mat2_load_index *= within_mat2;

            s_mat1[mat1_row_within_block * block_K + mat1_col_within_block] =
                mat1_buffer[mat1_load_index] * within_mat1;
            s_mat2[mat2_row_within_block * block_N + mat2_col_within_block] =
                mat2_buffer[mat2_load_index] * within_mat2;
        }

        mat1_pos += block_K;
        mat2_pos += block_K * N;

        __syncthreads();

        // Go through common dimensions of block (across row of mat1 and down col of mat2)
#pragma unroll 8
        for (int block_common_index = 0; block_common_index < block_K; block_common_index++) {
            // Now this thread will accumulate the block_K x block_K results from shared memory
#pragma unroll 8
            for (int result_index_row = 0; result_index_row < block_K; result_index_row++) {
#pragma unroll 8
                for (int result_index_col = 0; result_index_col < block_K; result_index_col++) {
                    thread_results[result_index_row * block_K + result_index_col] +=
                        s_mat1[(out_block_row * block_K + result_index_row) * block_K + block_common_index] *
                        s_mat2[(block_common_index * block_N) + (out_block_col * block_K + result_index_col)];
                }
            }
        }
        __syncthreads();
    }

    // Write results with bounds checking
    const int out_index_row = block_row * block_M + out_block_row * block_K;
    const int out_index_col = block_col * block_N + out_block_col * block_K;

#pragma unroll 8
    for (int i = 0; i < block_K; i++) {
#pragma unroll 8
        for (int j = 0; j < block_K; j++) {
            if (out_index_row + i < M && out_index_col + j < N) {
                out_buffer[(out_index_row + i) * N + out_index_col + j] = thread_results[i * block_K + j];
            }
        }
    }
}

size_t cuda_matrix_multiply(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat2_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_mat2_buffer = get_matrix_gpu_address(mat2_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    // const int THREADS_PER_BLOCK_X = 32;
    // const int THREADS_PER_BLOCK_Y = 32;

    // dim3 block_dim(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1);
    // dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // // Run the kernels
    // matrix_multiply_kernel_3<<<grid_dim, block_di>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);

    // V4 launch
    const int M = mat1_rows;
    const int N = mat2_cols;
    const int K = mat1_cols;

    const int THREADS_PER_BLOCK_X = 32;
    const int THREADS_PER_BLOCK_Y = 8;

    dim3 block_dim(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
    dim3 grid_dim((N + 128 - 1) / 128, (M + 128 - 1) / 128, 1);
    matrix_multiply_kernel_5<128, 128, 8><<<grid_dim, block_dim, 0, get_stream()>>>(M, N, K, gpu_mat1_buffer, gpu_mat2_buffer, gpu_out_buffer);

    // CUBLAS version (for comparison to mine)
    // float alpha = 1.0;
    // float beta = 0.0;
    // hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, mat2_cols, mat1_rows, mat1_cols, &alpha, gpu_mat2_buffer, mat2_cols, gpu_mat1_buffer, mat1_cols, &beta, gpu_out_buffer, mat2_cols);

    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void add_vector_to_columns_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] + mat2[i][0]

        int mat1_index = tidY * mat1_cols + tidX;
        int mat2_index = tidY;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] + mat2_buffer[mat2_index];
    }
}

__global__ void add_vector_to_rows_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] + mat2[0][j]

        int mat1_index = tidY * mat1_cols + tidX;
        int mat2_index = tidX;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] + mat2_buffer[mat2_index];
    }
}

size_t cuda_add_vector(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Determine orientation
    bool is_column_vector = (mat2_cols == 1 && mat2_rows == mat1_rows);

    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_mat2_buffer = get_matrix_gpu_address(mat2_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    if (is_column_vector) {
        add_vector_to_columns_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    } else {
        add_vector_to_rows_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    }
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void divide_by_column_vector_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] / mat2[i][0]

        int mat1_index = tidY * mat1_cols + tidX;
        int mat2_index = tidY;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] / mat2_buffer[mat2_index];
    }
}

__global__ void divide_by_row_vector_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] / mat2[0][j]

        int mat1_index = tidY * mat1_cols + tidX;
        int mat2_index = tidX;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] / mat2_buffer[mat2_index];
    }
}

size_t cuda_divide_by_vector(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Determine orientation
    bool is_column_vector = (mat2_cols == 1 && mat2_rows == mat1_rows);

    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_mat2_buffer = get_matrix_gpu_address(mat2_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    if (is_column_vector) {
        divide_by_column_vector_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    } else {
        divide_by_row_vector_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    }
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void element_sqrt_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = sqrt(mat1[i][j])

        int index = tidY * out_cols + tidX;
        out_buffer[index] = sqrt(mat1_buffer[index]);
    }
}

size_t cuda_element_sqrt(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    element_sqrt_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Each block handles one matrix
__global__ void cuda_element_sqrt_packed_kernel(float** mat_buffers, float** out_buffers, int mat_rows, int mat_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;
            out_buffer[index] = sqrt(mat_buffer[index]);
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}

void cuda_element_sqrt_packed(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Grab device pointers and put in pinned memory to upload as device array
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(2);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[1];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_element_sqrt_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, gpu_out_buffers_dp, mat_rows, mat_cols);
    gpuErrchk(hipPeekAtLastError());
}

void cuda_element_sqrt_packed_inplace(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    // Grab device pointers and put in pinned memory to upload as device array
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(1);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_element_sqrt_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, gpu_mat_buffers_dp, mat_rows, mat_cols);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void element_exp_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = exp(mat1[i][j])

        int index = tidY * out_cols + tidX;
        out_buffer[index] = exp(mat1_buffer[index]);
    }
}

size_t cuda_element_exp(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    element_exp_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Each block handles one matrix
__global__ void cuda_element_exp_packed_kernel(float** mat_buffers, float** out_buffers, int mat_rows, int mat_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;
            out_buffer[index] = exp(mat_buffer[index]);
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}

void cuda_element_exp_packed(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Grab device pointers and put in pinned memory to upload as device array
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(2);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[1];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_element_exp_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, gpu_out_buffers_dp, mat_rows, mat_cols);
    gpuErrchk(hipPeekAtLastError());
}

void cuda_element_exp_packed_inplace(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    // Grab device pointers and put in pinned memory to upload as device array
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(1);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_element_exp_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, gpu_mat_buffers_dp, mat_rows, mat_cols);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void element_ReLU_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = x if x>0 else 0

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] > 0 ? mat1_buffer[index] : 0.0;
    }
}

size_t cuda_element_ReLU(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    element_ReLU_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Each block handles one matrix
__global__ void cuda_element_ReLU_packed_kernel(float** mat_buffers, float** out_buffers, int mat_rows, int mat_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;
            out_buffer[index] = mat_buffer[index] > 0 ? mat_buffer[index] : 0.0;
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}

void cuda_element_ReLU_packed(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Grab device pointers and put in pinned memory to upload as device array
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(2);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[1];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_element_ReLU_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, gpu_out_buffers_dp, mat_rows, mat_cols);
    gpuErrchk(hipPeekAtLastError());
}

void cuda_element_ReLU_packed_inplace(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    // Grab device pointers and put in pinned memory to upload as device array
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(1);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_element_ReLU_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, gpu_mat_buffers_dp, mat_rows, mat_cols);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void element_ReLU_prime_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = x if x>0 else 1

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] > 0.0 ? 1.0 : 0.0;
    }
}

size_t cuda_element_ReLU_prime(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    element_ReLU_prime_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Each block handles one matrix
__global__ void cuda_element_ReLU_prime_packed_kernel(float** mat_buffers, float** out_buffers, int mat_rows, int mat_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < mat_rows) {
        while (tidX < mat_cols) {
            const int index = tidY * mat_cols + tidX;
            out_buffer[index] = mat_buffer[index] > 0.0 ? 1.0 : 0.0;
            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}

void cuda_element_ReLU_prime_packed(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Grab device pointers and put in pinned memory to upload as device array
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(2);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[1];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_element_ReLU_prime_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, gpu_out_buffers_dp, mat_rows, mat_cols);
    gpuErrchk(hipPeekAtLastError());
}

void cuda_element_ReLU_prime_packed_inplace(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    // Grab device pointers and put in pinned memory to upload as device array
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(2);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_element_ReLU_prime_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, gpu_mat_buffers_dp, mat_rows, mat_cols);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void sum_rows_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][0] = sum (mat1[i][:])

        float row_sum = 0.0;
        int mat1_row_start_index = tidY * mat1_cols;
        for (int i = 0; i < mat1_cols; i++) {
            int mat1_index = mat1_row_start_index + i;
            row_sum += mat1_buffer[mat1_index];
        }

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = row_sum;
    }
}

size_t cuda_sum_rows(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = 1;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    sum_rows_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void sum_columns_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[0][j] = sum (mat1[:][j])

        float col_sum = 0.0;
        for (int i = 0; i < mat1_rows; i++) {
            int mat1_index = tidX + i * mat1_cols;
            col_sum += mat1_buffer[mat1_index];
        }

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = col_sum;
    }
}

size_t cuda_sum_columns(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = 1;
    int out_cols = mat1_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    sum_columns_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void transpose_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[j][i]

        int mat1_index = tidX * mat1_cols + tidY;

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = mat1_buffer[mat1_index];
    }
}

size_t cuda_transpose(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = mat1_cols;
    int out_cols = mat1_rows;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    transpose_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void cuda_max_pool_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols, float* max_bitmask) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // For each 2x2 area pick the maximum value
        // We will mem coalesce by getting first two in row 1
        // Then next 2 in row2

        int block_start_row = tidY * 2;
        int block_start_col = tidX * 2;
        int block_start = block_start_row * mat1_cols + block_start_col;

        // bool block_00_oob = false;
        bool block_01_oob = (block_start_col + 1) >= mat1_cols;
        bool block_10_oob = (block_start_row + 1) >= mat1_rows;
        bool block_11_oob = block_01_oob || block_10_oob;

        // Unique small values to ensure bitmask is written once
        const float small_float_1 = -1e30;  // Should probably use FLT_MIN but language server no like it
        const float small_float_2 = -1e31;
        const float small_float_3 = -1e32;

        // TODO: Use bit operations instead of ternary (it's faster idk why the compiler can't figure it out)
        float block_00 = mat1_buffer[block_start];
        float block_01 = block_01_oob ? small_float_1 : mat1_buffer[block_start + 1];
        block_start += mat1_cols;
        float block_10 = block_10_oob ? small_float_2 : mat1_buffer[block_start];
        float block_11 = block_11_oob ? small_float_3 : mat1_buffer[block_start + 1];

        float result = max(max(block_00, block_01), max(block_10, block_11));

        // Set bitmask
        max_bitmask[block_start_row * mat1_cols + block_start_col] = (float)(result == block_00);
        if (!block_01_oob) {
            max_bitmask[block_start_row * mat1_cols + block_start_col + 1] = (float)(result == block_01);
        }
        if (!block_10_oob) {
            max_bitmask[(block_start_row + 1) * mat1_cols + block_start_col] = (float)(result == block_10);
        }
        if (!block_11_oob) {
            max_bitmask[(block_start_row + 1) * mat1_cols + block_start_col + 1] = (float)(result == block_11);
        }

        // Write maxpool result
        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = result;
    }
}

// 2x2 since other reduction sizes are not really used
Tuple cuda_max_pool(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = mat1_rows / 2 + mat1_rows % 2;
    int out_cols = mat1_cols / 2 + mat1_cols % 2;
    size_t out_mat_id = register_matrix(out_rows, out_cols);
    size_t max_bitmask = register_matrix(mat1_rows, mat1_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);
    float* gpu_max_bitmask = get_matrix_gpu_address(max_bitmask);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    cuda_max_pool_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols, gpu_max_bitmask);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return Tuple{out_mat_id, max_bitmask};
}

// Each block handles one matrix
__global__ void cuda_max_pool_packed_kernel(float** mat_buffers, float** out_buffers, float** max_bitmasks, int mat_rows, int mat_cols, int out_rows, int out_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];
    float* max_bitmask = max_bitmasks[current_matrix];

    // The work will be split among threads in the block
    while (tidY < out_rows) {
        while (tidX < out_cols) {
            // For each 2x2 area pick the maximum value
            // We will mem coalesce by getting first two in row 1
            // Then next 2 in row2

            int block_start_row = tidY * 2;
            int block_start_col = tidX * 2;
            int block_start = block_start_row * mat_cols + block_start_col;

            // bool block_00_oob = false;
            bool block_01_oob = (block_start_col + 1) >= mat_cols;
            bool block_10_oob = (block_start_row + 1) >= mat_rows;
            bool block_11_oob = block_01_oob || block_10_oob;

            // Unique small values to ensure bitmask is written once
            const float small_float_1 = -1e30;  // Should probably use FLT_MIN but language server no like it
            const float small_float_2 = -1e31;
            const float small_float_3 = -1e32;

            // TODO: Use bit operations instead of ternary (it's faster idk why the compiler can't figure it out)
            float block_00 = mat_buffer[block_start];
            float block_01 = block_01_oob ? small_float_1 : mat_buffer[block_start + 1];
            block_start += mat_cols;
            float block_10 = block_10_oob ? small_float_2 : mat_buffer[block_start];
            float block_11 = block_11_oob ? small_float_3 : mat_buffer[block_start + 1];

            float result = max(max(block_00, block_01), max(block_10, block_11));

            // Set bitmask
            max_bitmask[block_start_row * mat_cols + block_start_col] = (float)(result == block_00);
            if (!block_01_oob) {
                max_bitmask[block_start_row * mat_cols + block_start_col + 1] = (float)(result == block_01);
            }
            if (!block_10_oob) {
                max_bitmask[(block_start_row + 1) * mat_cols + block_start_col] = (float)(result == block_10);
            }
            if (!block_11_oob) {
                max_bitmask[(block_start_row + 1) * mat_cols + block_start_col + 1] = (float)(result == block_11);
            }

            // Write maxpool result
            int output_index = tidY * out_cols + tidX;
            out_buffer[output_index] = result;
            // printf("Set result to %f at row %d col %d for mat #%d at %d\n", result, tidY, tidX, current_matrix, output_index);

            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}

void cuda_max_pool_packed(size_t* mat_ids, Tuple* out_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows / 2 + mat_rows % 2;
    int out_cols = mat_cols / 2 + mat_cols % 2;

    std::vector<size_t> out_mat_ids(num_matrices);
    std::vector<size_t> max_bitmask_ids(num_matrices);
    register_matrix_group(out_rows, out_cols, num_matrices, &out_mat_ids[0]);
    register_matrix_group(mat_rows, mat_cols, num_matrices, &max_bitmask_ids[0]);

    for (int i = 0; i < num_matrices; i++) {
        size_t out_mat_id = out_mat_ids[i];
        size_t max_bitmask = max_bitmask_ids[i];
        out_ids[i] = Tuple{out_mat_id, max_bitmask};
    }

    // Get gpu buffers to oeprate on
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_max_bitmasks_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_ids[i].a);
        pinned_max_bitmasks_ptr[i] = get_matrix_gpu_address(out_ids[i].b);
    }

    // Upload the pointers to a gpu array
    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(3);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[1];
    float** gpu_max_bitmasks_dp = (float**)kernel_arg_device_pointers[2];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_max_bitmasks_dp, pinned_max_bitmasks_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_max_pool_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, gpu_out_buffers_dp, gpu_max_bitmasks_dp, mat_rows, mat_cols, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void cuda_nearest_neighbor_2x_upsample_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    // Upsample by nearest neighbor
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i/2][j/2]
        int mat1_index = (tidY / 2) * mat1_cols + (tidX / 2);

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = mat1_buffer[mat1_index];
    }
}

// Odd upsample will leave out one row and one column from the upsampled matrix
size_t cuda_nearest_neighbor_2x_upsample(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, bool odd_upsample) {
    // Create output buffer
    int out_rows = mat1_rows * 2 - (int)odd_upsample;
    int out_cols = mat1_cols * 2 - (int)odd_upsample;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK_X = 16;
    const int THREADS_PER_BLOCK_Y = 16;

    dim3 block_dim(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);
    cuda_nearest_neighbor_2x_upsample_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Each block handles one matrix
__global__ void cuda_nearest_neighbor_2x_upsample_packed_kernel(float** mat_buffers, float** out_buffers, int mat_rows, int mat_cols, int out_rows, int out_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidY < out_rows) {
        while (tidX < out_cols) {
            // O[i][j] = mat[i/2][j/2]
            int mat_index = (tidY / 2) * mat_cols + (tidX / 2);
            int output_index = tidY * out_cols + tidX;

            out_buffer[output_index] = mat_buffer[mat_index];

            tidX += blockDim.x;
        }
        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}

void cuda_nearest_neighbor_2x_upsample_packed(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, bool odd_upsample) {
    // Create output buffer
    int out_rows = mat_rows * 2 - (int)odd_upsample;
    int out_cols = mat_cols * 2 - (int)odd_upsample;

    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Get the gpu buffers to operate on
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(2);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[1];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_nearest_neighbor_2x_upsample_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, gpu_out_buffers_dp, mat_rows, mat_cols, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void cuda_rotate_180_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    const int mat_length = mat1_rows * mat1_cols;

    if (tidX < mat_length) {
        // Rotating an array 180 means
        // Reversing the linearized array
        const int reversed_index = mat_length - tidX - 1;
        const float input = mat1_buffer[reversed_index];

        const int output_index = tidX;
        out_buffer[output_index] = input;
    }
}

size_t cuda_rotate_180(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    int out_length = out_rows * out_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK_X = 256;
    dim3 block_dim(THREADS_PER_BLOCK_X, 1, 1);
    dim3 grid_dim((out_length + block_dim.x - 1) / block_dim.x, 1, 1);

    // Run the kernels
    cuda_rotate_180_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Each block handles one matrix
__global__ void cuda_rotate_180_packed_kernel(float** mat_buffers, float** out_buffers, int mat_rows, int mat_cols, int out_rows, int out_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    const int mat_length = out_rows * out_cols;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among threads in the block
    while (tidX < mat_length) {
        // Rotating an array 180 means
        // Reversing the linearized array
        const int reversed_index = mat_length - tidX - 1;
        const float input = mat_buffer[reversed_index];

        int output_index = tidX;
        out_buffer[output_index] = input;

        tidX += blockDim.x;
    }
}

void cuda_rotate_180_packed(size_t* mat_ids, size_t* out_mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Get the gpu buffers to operate on
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(2);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[1];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK_X = 256;
    dim3 block_dim(THREADS_PER_BLOCK_X, 1, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_rotate_180_packed_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, gpu_out_buffers_dp, mat_rows, mat_cols, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());
}

// Naive implementation
__global__ void cuda_correlate_kernel_valid_1(float* mat1_buffer, int mat1_rows, int mat1_cols, float* kernel_buffer, int kernel_rows, int kernel_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = weighted sum of kernel with input, where kernel is kept within bounds of input
        float result = 0.0;
        const int kernel_top_left_row = tidY;
        const int kernel_top_left_col = tidX;

#pragma unroll 3
        for (int m = 0; m < kernel_rows; m++) {
#pragma unroll 3
            for (int n = 0; n < kernel_cols; n++) {
                const float mat1_val = mat1_buffer[(kernel_top_left_row + m) * mat1_cols + (kernel_top_left_col + n)];
                const float kernel_val = kernel_buffer[m * kernel_cols + n];
                result += mat1_val * kernel_val;
            }
        }

        const int out_index = tidY * out_cols + tidX;
        out_buffer[out_index] = result;
    }
}

// Partial Shared Memory implementation
template <const int block_x, const int block_y>
__global__ void cuda_correlate_kernel_valid_2(float* mat1_buffer, int mat1_rows, int mat1_cols, float* kernel_buffer, int kernel_rows, int kernel_cols, float* out_buffer, int out_rows, int out_cols) {
    // Create shared memory
    __shared__ float mat1_shared[block_x * block_y];

    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;
    int threadIdWithinBlock = threadIdx.y * blockDim.x + threadIdx.x;

    // Load data into shared memory
    mat1_shared[threadIdWithinBlock] = mat1_buffer[tidY * mat1_cols + tidX];
    __syncthreads();

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = weighted sum of kernel with input, where kernel is kept within bounds of input
        float result = 0.0;
        const int kernel_top_left_row = tidY;
        const int kernel_top_left_col = tidX;

#pragma unroll 3
        for (int m = 0; m < kernel_rows; m++) {
#pragma unroll 3
            for (int n = 0; n < kernel_cols; n++) {
                int index_in_shared = (threadIdx.y + m) * blockDim.x + (threadIdx.x + n);
                int index_in_global = (kernel_top_left_row + m) * mat1_cols + (kernel_top_left_col + n);
                bool in_shared_bounds = index_in_shared < (block_x * block_y);

                const float mat1_val = in_shared_bounds ? mat1_shared[index_in_shared] : mat1_buffer[index_in_global];
                const float kernel_val = kernel_buffer[m * kernel_cols + n];
                result += mat1_val * kernel_val;
            }
        }

        const int out_index = tidY * out_cols + tidX;
        out_buffer[out_index] = result;
    }
}

// Be careful, this needs to be optimized or your CNN will suffer
size_t cuda_correlate_valid(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t kernel_id, size_t kernel_rows, size_t kernel_cols) {
    // Create output buffer
    // Dimension of output is input - kernel + 1
    int out_rows = mat1_rows - kernel_rows + 1;
    int out_cols = mat1_cols - kernel_cols + 1;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_kernel_buffer = get_matrix_gpu_address(kernel_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    cuda_correlate_kernel_valid_1<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_kernel_buffer, kernel_rows, kernel_cols, gpu_out_buffer, out_rows, out_cols);
    // cuda_correlate_kernel_valid_2<THREADS_PER_BLOCK, THREADS_PER_BLOCK><<<grid_dim, block_di>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_kernel_buffer, kernel_rows, kernel_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Naive implementation
__global__ void cuda_correlate_kernel_same_1(float* mat1_buffer, int mat1_rows, int mat1_cols, float* kernel_buffer, int kernel_rows, int kernel_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = weighted sum of kernel with input, where kernel is centered at i,j

        float result = 0.0;
        const int apothem = kernel_rows / 2;
#pragma unroll 3
        for (int m = 0; m < kernel_rows; m++) {
#pragma unroll 3
            for (int n = 0; n < kernel_cols; n++) {
                int input_row = m - apothem + tidY;
                int input_col = n - apothem + tidX;
                bool input_row_in_bounds = input_row >= 0 && input_row < mat1_rows;
                bool input_col_in_bounds = input_col >= 0 && input_col < mat1_cols;

                if (input_row_in_bounds && input_col_in_bounds) {
                    const int curr_mat1_index = input_row * mat1_cols + input_col;
                    const int curr_kernel_index = m * kernel_cols + n;
                    result += mat1_buffer[curr_mat1_index] * kernel_buffer[curr_kernel_index];
                }
            }
        }

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = result;
    }
}

// correlation is zero-padded (Output is the same size as input)
// Expects odd size, square kernels ONLY
// Be careful, this needs to be optimized or your CNN will suffer
size_t cuda_correlate_same(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t kernel_id, size_t kernel_rows, size_t kernel_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_kernel_buffer = get_matrix_gpu_address(kernel_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    cuda_correlate_kernel_same_1<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_kernel_buffer, kernel_rows, kernel_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Naive implementation
__global__ void cuda_correlate_kernel_full_1(float* mat1_buffer, int mat1_rows, int mat1_cols, float* kernel_buffer, int kernel_rows, int kernel_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = weighted sum of kernel with input, where kernel is centered at i,j
        float result = 0.0;
        const int input_start_row = (-kernel_rows + 1) + tidY;
        const int input_start_col = (-kernel_cols + 1) + tidX;
        for (int m = 0; m < kernel_rows; m++) {
            for (int n = 0; n < kernel_cols; n++) {
                int input_row = input_start_row + m;
                int input_col = input_start_col + n;
                bool input_row_in_bounds = input_row >= 0 && input_row < mat1_rows;
                bool input_col_in_bounds = input_col >= 0 && input_col < mat1_cols;

                if (input_row_in_bounds && input_col_in_bounds) {
                    const int curr_mat1_index = input_row * mat1_cols + input_col;
                    const int curr_kernel_index = m * kernel_cols + n;
                    result += mat1_buffer[curr_mat1_index] * kernel_buffer[curr_kernel_index];
                }
            }
        }

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = result;
    }
}

// Be careful, this needs to be optimized or your CNN will suffer
size_t cuda_correlate_full(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t kernel_id, size_t kernel_rows, size_t kernel_cols) {
    // Create output buffer
    // Dimension of output is input + kernel - 1
    int out_rows = mat1_rows + kernel_rows - 1;
    int out_cols = mat1_cols + kernel_cols - 1;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_kernel_buffer = get_matrix_gpu_address(kernel_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    cuda_correlate_kernel_full_1<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_kernel_buffer, kernel_rows, kernel_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

size_t cuda_correlate(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t kernel_id, size_t kernel_rows, size_t kernel_cols, PaddingType padding_type) {
    if (padding_type == PaddingType::VALID) {
        return cuda_correlate_valid(mat1_id, mat1_rows, mat1_cols, kernel_id, kernel_rows, kernel_cols);
    } else if (padding_type == PaddingType::SAME) {
        return cuda_correlate_same(mat1_id, mat1_rows, mat1_cols, kernel_id, kernel_rows, kernel_cols);
    } else if (padding_type == PaddingType::FULL) {
        return cuda_correlate_full(mat1_id, mat1_rows, mat1_cols, kernel_id, kernel_rows, kernel_cols);
    }
}

// Each block handles one matrix
__global__ void cuda_correlate_kernel_packed_valid_1(float** mat_buffers, int num_matrices, int mat_rows, int mat_cols, float** kernel_buffers, int kernel_rows, int kernel_cols, float** out_buffers, int out_rows, int out_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    const float* kernel_buffer = kernel_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among the threads in the block
    // Each thread will work until of tidX or tidY is out of bounds
    while (tidY < out_rows) {
        while (tidX < out_cols) {
            // Now perform correlation at this location
            float result = 0.0;
            const int kernel_top_left_row = tidY;
            const int kernel_top_left_col = tidX;

#pragma unroll 3
            for (int m = 0; m < kernel_rows; m++) {
#pragma unroll 3
                for (int n = 0; n < kernel_cols; n++) {
                    const float mat1_val = mat_buffer[(kernel_top_left_row + m) * mat_cols + (kernel_top_left_col + n)];
                    const float kernel_val = kernel_buffer[m * kernel_cols + n];
                    result += mat1_val * kernel_val;
                }
            }

            const int out_index = tidY * out_cols + tidX;
            out_buffer[out_index] = result;
            tidX += blockDim.x;
        }

        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}

// Should be used when you have a lot of small matrices to convolve
void cuda_correlate_valid_packed(size_t* matrices_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, size_t* kernel_ids, size_t kernel_rows, size_t kernel_cols, size_t* out_mat_ids) {
    // Create output buffer
    // Dimension of output is input - kernel + 1
    int out_rows = mat_rows - kernel_rows + 1;
    int out_cols = mat_cols - kernel_cols + 1;

    // Register output matrices
    // Register large buffer
    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Get the gpu buffers to operate on
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_kernel_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(matrices_ids[i]);
        pinned_kernel_buffers_ptr[i] = get_matrix_gpu_address(kernel_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(3);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_kernel_buffers_dp = (float**)kernel_arg_device_pointers[1];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[2];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_kernel_buffers_dp, pinned_kernel_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 8;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_correlate_kernel_packed_valid_1<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, num_matrices, mat_rows, mat_cols, gpu_kernel_buffers_dp, kernel_rows, kernel_cols, gpu_out_buffers_dp, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());
}

// Each block handles one correlation
__global__ void cuda_correlate_kernel_packed_same_1(float** mat_buffers, int num_matrices, int mat_rows, int mat_cols, float** kernel_buffers, int kernel_rows, int kernel_cols, float** out_buffers, int out_rows, int out_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    const float* kernel_buffer = kernel_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among the threads in the block
    // Each thread will work until of tidX or tidY is out of bounds
    while (tidY < out_rows) {
        while (tidX < out_cols) {
            float result = 0.0;
            const int apothem = kernel_rows / 2;
#pragma unroll 3
            for (int m = 0; m < kernel_rows; m++) {
#pragma unroll 3
                for (int n = 0; n < kernel_cols; n++) {
                    int input_row = m - apothem + tidY;
                    int input_col = n - apothem + tidX;
                    bool input_row_in_bounds = input_row >= 0 && input_row < mat_rows;
                    bool input_col_in_bounds = input_col >= 0 && input_col < mat_cols;

                    if (input_row_in_bounds && input_col_in_bounds) {
                        const int curr_mat1_index = input_row * mat_cols + input_col;
                        const int curr_kernel_index = m * kernel_cols + n;
                        result += mat_buffer[curr_mat1_index] * kernel_buffer[curr_kernel_index];
                    }
                }
            }

            int output_index = tidY * out_cols + tidX;
            out_buffer[output_index] = result;
            tidX += blockDim.x;
        }

        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}

// Should be used when you have a lot of small matrices to convolve
void cuda_correlate_same_packed(size_t* matrices_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, size_t* kernel_ids, size_t kernel_rows, size_t kernel_cols, size_t* out_mat_ids) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    // Register output matrices
    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Get the gpu buffers to operate on
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_kernel_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(matrices_ids[i]);
        pinned_kernel_buffers_ptr[i] = get_matrix_gpu_address(kernel_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(3);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_kernel_buffers_dp = (float**)kernel_arg_device_pointers[1];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[2];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_kernel_buffers_dp, pinned_kernel_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 8;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_correlate_kernel_packed_same_1<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, num_matrices, mat_rows, mat_cols, gpu_kernel_buffers_dp, kernel_rows, kernel_cols, gpu_out_buffers_dp, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());
}

// Each block handles one correlation
__global__ void cuda_correlate_kernel_packed_full_1(float** mat_buffers, int num_matrices, int mat_rows, int mat_cols, float** kernel_buffers, int kernel_rows, int kernel_cols, float** out_buffers, int out_rows, int out_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    const float* kernel_buffer = kernel_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among the threads in the block
    // Each thread will work until of tidX or tidY is out of bounds
    while (tidY < out_rows) {
        while (tidX < out_cols) {
            // O[i][j] = weighted sum of kernel with input, where kernel is centered at i,j
            float result = 0.0;
            const int input_start_row = (-kernel_rows + 1) + tidY;
            const int input_start_col = (-kernel_cols + 1) + tidX;
            for (int m = 0; m < kernel_rows; m++) {
                for (int n = 0; n < kernel_cols; n++) {
                    int input_row = input_start_row + m;
                    int input_col = input_start_col + n;
                    bool input_row_in_bounds = input_row >= 0 && input_row < mat_rows;
                    bool input_col_in_bounds = input_col >= 0 && input_col < mat_cols;

                    if (input_row_in_bounds && input_col_in_bounds) {
                        const int curr_mat1_index = input_row * mat_cols + input_col;
                        const int curr_kernel_index = m * kernel_cols + n;
                        result += mat_buffer[curr_mat1_index] * kernel_buffer[curr_kernel_index];
                    }
                }
            }

            int output_index = tidY * out_cols + tidX;
            out_buffer[output_index] = result;
            tidX += blockDim.x;
        }

        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}

// Should be used when you have a lot of small matrices to convolve
void cuda_correlate_full_packed(size_t* matrices_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, size_t* kernel_ids, size_t kernel_rows, size_t kernel_cols, size_t* out_mat_ids) {
    // Create output buffer
    // Dimension of output is input + kernel - 1
    int out_rows = mat_rows + kernel_rows - 1;
    int out_cols = mat_cols + kernel_cols - 1;

    // Register output matrices
    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Get the gpu buffers to operate on
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_kernel_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(matrices_ids[i]);
        pinned_kernel_buffers_ptr[i] = get_matrix_gpu_address(kernel_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    // Upload the pointers to a gpu array
    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(3);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_kernel_buffers_dp = (float**)kernel_arg_device_pointers[1];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[2];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_kernel_buffers_dp, pinned_kernel_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 8;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_correlate_kernel_packed_full_1<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, num_matrices, mat_rows, mat_cols, gpu_kernel_buffers_dp, kernel_rows, kernel_cols, gpu_out_buffers_dp, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());
}
void cuda_correlate_packed(size_t* mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, size_t* kernel_ids, size_t kernel_rows, size_t kernel_cols, size_t* out_ids, PaddingType padding_type) {
    if (padding_type == PaddingType::VALID) {
        return cuda_correlate_valid_packed(mat_ids, num_matrices, mat_rows, mat_cols, kernel_ids, kernel_rows, kernel_cols, out_ids);
    } else if (padding_type == PaddingType::SAME) {
        return cuda_correlate_same_packed(mat_ids, num_matrices, mat_rows, mat_cols, kernel_ids, kernel_rows, kernel_cols, out_ids);
    } else if (padding_type == PaddingType::FULL) {
        return cuda_correlate_full_packed(mat_ids, num_matrices, mat_rows, mat_cols, kernel_ids, kernel_rows, kernel_cols, out_ids);
    }
}

// Naive implementation
__global__ void cuda_convolve_kernel_valid_1(float* mat1_buffer, int mat1_rows, int mat1_cols, float* kernel_buffer, int kernel_rows, int kernel_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;
    const int kernel_length = kernel_rows * kernel_cols;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = weighted sum of kernel with input, where kernel is kept within bounds of input
        float result = 0.0;
        const int kernel_top_left_row = tidY;
        const int kernel_top_left_col = tidX;

#pragma unroll 3
        for (int m = 0; m < kernel_rows; m++) {
#pragma unroll 3
            for (int n = 0; n < kernel_cols; n++) {
                const float mat1_val = mat1_buffer[(kernel_top_left_row + m) * mat1_cols + (kernel_top_left_col + n)];
                const int rotated_kernel_position = kernel_length - (m * kernel_cols + n) - 1;  // Equivalent to reversing linearized kernel
                const float kernel_val = kernel_buffer[rotated_kernel_position];
                result += mat1_val * kernel_val;
            }
        }

        const int out_index = tidY * out_cols + tidX;
        out_buffer[out_index] = result;
    }
}

// Be careful, this needs to be optimized or your CNN will suffer
size_t cuda_convolve_valid(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t kernel_id, size_t kernel_rows, size_t kernel_cols) {
    // Create output buffer
    // Dimension of output is input - kernel + 1
    int out_rows = mat1_rows - kernel_rows + 1;
    int out_cols = mat1_cols - kernel_cols + 1;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_kernel_buffer = get_matrix_gpu_address(kernel_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    cuda_convolve_kernel_valid_1<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_kernel_buffer, kernel_rows, kernel_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Naive implementation
__global__ void cuda_convolve_kernel_same_1(float* mat1_buffer, int mat1_rows, int mat1_cols, float* kernel_buffer, int kernel_rows, int kernel_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;
    const int kernel_length = kernel_rows * kernel_cols;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = weighted sum of kernel with input, where kernel is centered at i,j

        float result = 0.0;
        const int apothem = kernel_rows / 2;
#pragma unroll 3
        for (int m = 0; m < kernel_rows; m++) {
#pragma unroll 3
            for (int n = 0; n < kernel_cols; n++) {
                int input_row = m - apothem + tidY;
                int input_col = n - apothem + tidX;
                bool input_row_in_bounds = input_row >= 0 && input_row < mat1_rows;
                bool input_col_in_bounds = input_col >= 0 && input_col < mat1_cols;

                if (input_row_in_bounds && input_col_in_bounds) {
                    const int curr_mat1_index = input_row * mat1_cols + input_col;
                    const int rotated_kernel_position = kernel_length - (m * kernel_cols + n) - 1;  // Equivalent to reversing linearized kernel
                    result += mat1_buffer[curr_mat1_index] * kernel_buffer[rotated_kernel_position];
                }
            }
        }

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = result;
    }
}

// correlation is zero-padded (Output is the same size as input)
// Expects odd size, square kernels ONLY
// Be careful, this needs to be optimized or your CNN will suffer
size_t cuda_convolve_same(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t kernel_id, size_t kernel_rows, size_t kernel_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_kernel_buffer = get_matrix_gpu_address(kernel_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    cuda_convolve_kernel_same_1<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_kernel_buffer, kernel_rows, kernel_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Naive implementation
__global__ void cuda_convolve_kernel_full_1(float* mat1_buffer, int mat1_rows, int mat1_cols, float* kernel_buffer, int kernel_rows, int kernel_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;
    const int kernel_length = kernel_rows * kernel_cols;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = weighted sum of kernel with input, where kernel is centered at i,j
        float result = 0.0;
        const int input_start_row = (-kernel_rows + 1) + tidY;
        const int input_start_col = (-kernel_cols + 1) + tidX;
        for (int m = 0; m < kernel_rows; m++) {
            for (int n = 0; n < kernel_cols; n++) {
                int input_row = input_start_row + m;
                int input_col = input_start_col + n;
                bool input_row_in_bounds = input_row >= 0 && input_row < mat1_rows;
                bool input_col_in_bounds = input_col >= 0 && input_col < mat1_cols;

                if (input_row_in_bounds && input_col_in_bounds) {
                    const int curr_mat1_index = input_row * mat1_cols + input_col;
                    const int rotated_kernel_position = kernel_length - (m * kernel_cols + n) - 1;  // Equivalent to reversing linearized kernel
                    result += mat1_buffer[curr_mat1_index] * kernel_buffer[rotated_kernel_position];
                }
            }
        }

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = result;
    }
}

// Be careful, this needs to be optimized or your CNN will suffer
size_t cuda_convolve_full(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t kernel_id, size_t kernel_rows, size_t kernel_cols) {
    // Create output buffer
    // Dimension of output is input + kernel - 1
    int out_rows = mat1_rows + kernel_rows - 1;
    int out_cols = mat1_cols + kernel_cols - 1;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = get_matrix_gpu_address(mat1_id);
    float* gpu_kernel_buffer = get_matrix_gpu_address(kernel_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    cuda_convolve_kernel_full_1<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_kernel_buffer, kernel_rows, kernel_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

size_t cuda_convolve(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t kernel_id, size_t kernel_rows, size_t kernel_cols, PaddingType padding_type) {
    if (padding_type == PaddingType::VALID) {
        return cuda_convolve_valid(mat1_id, mat1_rows, mat1_cols, kernel_id, kernel_rows, kernel_cols);
    } else if (padding_type == PaddingType::SAME) {
        return cuda_convolve_same(mat1_id, mat1_rows, mat1_cols, kernel_id, kernel_rows, kernel_cols);
    } else if (padding_type == PaddingType::FULL) {
        return cuda_convolve_full(mat1_id, mat1_rows, mat1_cols, kernel_id, kernel_rows, kernel_cols);
    }
}

// Each block handles one matrix
__global__ void cuda_convolve_kernel_packed_valid_1(float** mat_buffers, int num_matrices, int mat_rows, int mat_cols, float** kernel_buffers, int kernel_rows, int kernel_cols, float** out_buffers, int out_rows, int out_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;
    const int kernel_length = kernel_rows * kernel_cols;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    const float* kernel_buffer = kernel_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among the threads in the block
    // Each thread will work until of tidX or tidY is out of bounds
    while (tidY < out_rows) {
        while (tidX < out_cols) {
            // Now perform correlation at this location
            float result = 0.0;
            const int kernel_top_left_row = tidY;
            const int kernel_top_left_col = tidX;

#pragma unroll 3
            for (int m = 0; m < kernel_rows; m++) {
#pragma unroll 3
                for (int n = 0; n < kernel_cols; n++) {
                    const float mat1_val = mat_buffer[(kernel_top_left_row + m) * mat_cols + (kernel_top_left_col + n)];
                    const int rotated_kernel_position = kernel_length - (m * kernel_cols + n) - 1;  // Equivalent to reversing linearized kernel
                    const float kernel_val = kernel_buffer[rotated_kernel_position];
                    result += mat1_val * kernel_val;
                }
            }

            const int out_index = tidY * out_cols + tidX;
            out_buffer[out_index] = result;
            tidX += blockDim.x;
        }

        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}

// Should be used when you have a lot of small matrices to convolve
void cuda_convolve_valid_packed(size_t* matrices_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, size_t* kernel_ids, size_t kernel_rows, size_t kernel_cols, size_t* out_mat_ids) {
    // Create output buffer
    // Dimension of output is input - kernel + 1
    int out_rows = mat_rows - kernel_rows + 1;
    int out_cols = mat_cols - kernel_cols + 1;

    // Register output matrices
    // Register large buffer
    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Get the gpu buffers to operate on
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_kernel_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(matrices_ids[i]);
        pinned_kernel_buffers_ptr[i] = get_matrix_gpu_address(kernel_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    // Upload the pointers to a gpu array
    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(3);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_kernel_buffers_dp = (float**)kernel_arg_device_pointers[1];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[2];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_kernel_buffers_dp, pinned_kernel_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 8;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_convolve_kernel_packed_valid_1<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, num_matrices, mat_rows, mat_cols, gpu_kernel_buffers_dp, kernel_rows, kernel_cols, gpu_out_buffers_dp, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());
}

// Each block handles one correlation
__global__ void cuda_convolve_kernel_packed_same_1(float** mat_buffers, int num_matrices, int mat_rows, int mat_cols, float** kernel_buffers, int kernel_rows, int kernel_cols, float** out_buffers, int out_rows, int out_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;
    const int kernel_length = kernel_rows * kernel_cols;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    const float* kernel_buffer = kernel_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among the threads in the block
    // Each thread will work until of tidX or tidY is out of bounds
    while (tidY < out_rows) {
        while (tidX < out_cols) {
            float result = 0.0;
            const int apothem = kernel_rows / 2;
#pragma unroll 3
            for (int m = 0; m < kernel_rows; m++) {
#pragma unroll 3
                for (int n = 0; n < kernel_cols; n++) {
                    int input_row = m - apothem + tidY;
                    int input_col = n - apothem + tidX;
                    bool input_row_in_bounds = input_row >= 0 && input_row < mat_rows;
                    bool input_col_in_bounds = input_col >= 0 && input_col < mat_cols;

                    if (input_row_in_bounds && input_col_in_bounds) {
                        const int curr_mat1_index = input_row * mat_cols + input_col;
                        const int rotated_kernel_position = kernel_length - (m * kernel_cols + n) - 1;  // Equivalent to reversing linearized kernel
                        result += mat_buffer[curr_mat1_index] * kernel_buffer[rotated_kernel_position];
                    }
                }
            }

            int output_index = tidY * out_cols + tidX;
            out_buffer[output_index] = result;
            tidX += blockDim.x;
        }

        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}

// Should be used when you have a lot of small matrices to convolve
void cuda_convolve_same_packed(size_t* matrices_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, size_t* kernel_ids, size_t kernel_rows, size_t kernel_cols, size_t* out_mat_ids) {
    // Create output buffer
    int out_rows = mat_rows;
    int out_cols = mat_cols;

    // Register output matrices
    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Get the gpu buffers to operate on
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_kernel_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(matrices_ids[i]);
        pinned_kernel_buffers_ptr[i] = get_matrix_gpu_address(kernel_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    // Upload the pointers to a gpu array
    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(3);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_kernel_buffers_dp = (float**)kernel_arg_device_pointers[1];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[2];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_kernel_buffers_dp, pinned_kernel_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 8;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_convolve_kernel_packed_same_1<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, num_matrices, mat_rows, mat_cols, gpu_kernel_buffers_dp, kernel_rows, kernel_cols, gpu_out_buffers_dp, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());
}

// Each block handles one correlation
__global__ void cuda_convolve_kernel_packed_full_1(float** mat_buffers, int num_matrices, int mat_rows, int mat_cols, float** kernel_buffers, int kernel_rows, int kernel_cols, float** out_buffers, int out_rows, int out_cols) {
    const int current_matrix = blockIdx.x;
    int tidX = threadIdx.x;
    int tidY = threadIdx.y;
    const int kernel_length = kernel_rows * kernel_cols;

    // Grab the buffers
    const float* mat_buffer = mat_buffers[current_matrix];
    const float* kernel_buffer = kernel_buffers[current_matrix];
    float* out_buffer = out_buffers[current_matrix];

    // The work will be split among the threads in the block
    // Each thread will work until of tidX or tidY is out of bounds
    while (tidY < out_rows) {
        while (tidX < out_cols) {
            // O[i][j] = weighted sum of kernel with input, where kernel is centered at i,j
            float result = 0.0;
            const int input_start_row = (-kernel_rows + 1) + tidY;
            const int input_start_col = (-kernel_cols + 1) + tidX;
            for (int m = 0; m < kernel_rows; m++) {
                for (int n = 0; n < kernel_cols; n++) {
                    int input_row = input_start_row + m;
                    int input_col = input_start_col + n;
                    bool input_row_in_bounds = input_row >= 0 && input_row < mat_rows;
                    bool input_col_in_bounds = input_col >= 0 && input_col < mat_cols;

                    if (input_row_in_bounds && input_col_in_bounds) {
                        const int curr_mat1_index = input_row * mat_cols + input_col;
                        const int rotated_kernel_position = kernel_length - (m * kernel_cols + n) - 1;  // Equivalent to reversing linearized kernel
                        result += mat_buffer[curr_mat1_index] * kernel_buffer[rotated_kernel_position];
                    }
                }
            }

            int output_index = tidY * out_cols + tidX;
            out_buffer[output_index] = result;
            tidX += blockDim.x;
        }

        tidX = threadIdx.x;
        tidY += blockDim.y;
    }
}

// Should be used when you have a lot of small matrices to convolve
void cuda_convolve_full_packed(size_t* matrices_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, size_t* kernel_ids, size_t kernel_rows, size_t kernel_cols, size_t* out_mat_ids) {
    // Create output buffer
    // Dimension of output is input + kernel - 1
    int out_rows = mat_rows + kernel_rows - 1;
    int out_cols = mat_cols + kernel_cols - 1;

    // Register output matrices
    register_matrix_group(out_rows, out_cols, num_matrices, out_mat_ids);

    // Get the gpu buffers to operate on
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_kernel_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    float** pinned_out_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);

    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(matrices_ids[i]);
        pinned_kernel_buffers_ptr[i] = get_matrix_gpu_address(kernel_ids[i]);
        pinned_out_buffers_ptr[i] = get_matrix_gpu_address(out_mat_ids[i]);
    }

    // Upload the pointers to a gpu array
    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(3);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];
    float** gpu_kernel_buffers_dp = (float**)kernel_arg_device_pointers[1];
    float** gpu_out_buffers_dp = (float**)kernel_arg_device_pointers[2];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_kernel_buffers_dp, pinned_kernel_buffers_ptr, sizeof(float*) * num_matrices);
    memory_manager_upload_from_pinned_buffer(gpu_out_buffers_dp, pinned_out_buffers_ptr, sizeof(float*) * num_matrices);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 8;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim(num_matrices, 1, 1);

    // Run the kernels
    cuda_convolve_kernel_packed_full_1<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, num_matrices, mat_rows, mat_cols, gpu_kernel_buffers_dp, kernel_rows, kernel_cols, gpu_out_buffers_dp, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());
}
void cuda_convolve_packed(size_t* mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, size_t* kernel_ids, size_t kernel_rows, size_t kernel_cols, size_t* out_ids, PaddingType padding_type) {
    if (padding_type == PaddingType::VALID) {
        return cuda_convolve_valid_packed(mat_ids, num_matrices, mat_rows, mat_cols, kernel_ids, kernel_rows, kernel_cols, out_ids);
    } else if (padding_type == PaddingType::SAME) {
        return cuda_convolve_same_packed(mat_ids, num_matrices, mat_rows, mat_cols, kernel_ids, kernel_rows, kernel_cols, out_ids);
    } else if (padding_type == PaddingType::FULL) {
        return cuda_convolve_full_packed(mat_ids, num_matrices, mat_rows, mat_cols, kernel_ids, kernel_rows, kernel_cols, out_ids);
    }
}

__global__ void cuda_img2col_valid(float** mat_buffers, int input_depth, int input_rows, int input_cols, int filter_depth, int filter_rows, int filter_cols, float* out_buffer, int out_rows, int out_cols) {
    const int tidX = blockDim.x * blockIdx.x + threadIdx.x;

    // This thread will handle one patch of the image, through all the kernels
    // This means each thread handle one column of the output
    const int number_of_patches = out_cols;
    const int current_patch = tidX;

    if (current_patch < number_of_patches) {
// Go through each of the kernels
#pragma unroll
        for (int curr_channel = 0; curr_channel < input_depth; curr_channel++) {
            const float* current_buffer = mat_buffers[curr_channel];

            // Now translate current_patch into the patch's top left corner
            const int kernel_top_left_row = current_patch / (input_cols - filter_cols + 1);
            const int kernel_top_left_col = current_patch % (input_cols - filter_cols + 1);
            const int base_output_row = curr_channel * filter_rows * filter_cols;
            const int output_col = current_patch;

// Now construct the patch
#pragma unroll 3
            for (int m = 0; m < filter_rows; m++) {
#pragma unroll 3
                for (int n = 0; n < filter_cols; n++) {
                    const float mat_val = current_buffer[(kernel_top_left_row + m) * input_cols + (kernel_top_left_col + n)];
                    const int output_index = base_output_row + m * filter_cols + n;
                    out_buffer[output_index * out_cols + output_col] = mat_val;
                }
            }
        }
    }
}

size_t cuda_img2col_valid(size_t* mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, size_t kernel_rows, size_t kernel_cols) {
    // Create output buffer
    const int kernel_count = num_matrices;  // num_matrices is the input depth
    const int out_rows = kernel_count * kernel_rows * kernel_cols;
    const int out_cols = (mat_rows - kernel_rows + 1) * (mat_cols - kernel_cols + 1);
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(size_t) * num_matrices);
    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
    }

    // Upload the pointers to a gpu array
    // Each allocation pair contains block_id, block_offset
    float** gpu_mat_buffers = (float**)get_device_kernel_args_pointers(1)[0];
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);

    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    // Let's have each threads handle its own patch between all the kernels
    // So we will calculate the number of patches == number of columns
    // Data access should be coalesced this way
    const int THREADS_PER_BLOCK = 1024;
    dim3 block_dim(THREADS_PER_BLOCK, 1, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, 1, 1);

    // Run the kernels
    cuda_img2col_valid<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers, num_matrices, mat_rows, mat_cols, kernel_count, kernel_rows, kernel_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Take an image and convert it to a matrix of columns based on patches (with specified padding) the filter makes of image
size_t cuda_img2col(size_t* mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, size_t kernel_rows, size_t kernel_cols, PaddingType padding_type) {
    if (padding_type == PaddingType::VALID) {
        return cuda_img2col_valid(mat_ids, num_matrices, mat_rows, mat_cols, kernel_rows, kernel_cols);
    } else if (padding_type == PaddingType::SAME) {
        return 0;
    } else if (padding_type == PaddingType::FULL) {
        return 0;
    }
}

__global__ void cuda_flatten_array_kernel(float** mat_buffers, int mat_rows, int mat_cols, float* out_buffer, int out_rows, int out_cols) {
    const int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    const int tidY = blockDim.y * blockIdx.y + threadIdx.y;
    const int output_index = tidY * out_cols + tidX;
    const int output_img_size = out_rows * out_cols;
    const int each_input_img_size = mat_rows * mat_cols;

    if (output_index < output_img_size) {
        // Grab the gpu buffer we are reffering to
        const int current_buffer_index = output_index / each_input_img_size;
        const float* current_buffer = mat_buffers[current_buffer_index];

        // Determine the pixel to copy
        const int current_buffer_pixel = output_index % each_input_img_size;

        // Write result
        out_buffer[output_index] = current_buffer[current_buffer_pixel];
    }
}

// Take n same_dimension matrices and flatten them into an array
size_t cuda_flatten_array(size_t* mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = 1;
    int out_cols = num_matrices * mat_rows * mat_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(1);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);

    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    dim3 block_dim(256, 1, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    cuda_flatten_array_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffers_dp, mat_rows, mat_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void cuda_unflatten_array_kernel(float* array_buffer, int arr_size, int mat_rows, int mat_cols, float** mat_buffers) {
    const int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    const int array_buffer_index = tidX;

    if (array_buffer_index < arr_size) {
        // Check which mat_buffer to write to
        const int mat_size = mat_rows * mat_cols;
        const int mat_buffer_index = array_buffer_index / mat_size;
        const int mat_buffer_pixel = array_buffer_index % mat_size;

        // Write result
        mat_buffers[mat_buffer_index][mat_buffer_pixel] = array_buffer[array_buffer_index];
    }
}

// Take an array and unflatten it into n same_dimension matrices.
void cuda_unflatten_array(size_t array_id, size_t arr_size, size_t mat_rows, size_t mat_cols, size_t* mat_ids) {
    int mat_size = mat_rows * mat_cols;
    int num_matrices = arr_size / mat_size;

    // Create the buffers for the matrices
    register_matrix_group(mat_rows, mat_cols, num_matrices, mat_ids);
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(1);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);

    // Get the flattened array
    float* gpu_array_buffer = get_matrix_gpu_address(array_id);

    // Kernel launch parameters
    dim3 block_dim(256, 1, 1);
    dim3 grid_dim((arr_size + block_dim.x - 1) / block_dim.x, 1, 1);

    // Run the kernels
    cuda_unflatten_array_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_array_buffer, arr_size, mat_rows, mat_cols, gpu_mat_buffers_dp);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void cuda_unflatten_array_strided_kernel(float* array_buffer, int arr_size, int num_matrices, int mat_rows, int mat_cols, float** mat_buffers) {
    const int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    const int array_buffer_index = tidX;

    if (array_buffer_index < arr_size) {
        // Check which mat_buffer to write to
        const int mat_buffer_index = array_buffer_index % num_matrices;
        const int mat_buffer_pixel = array_buffer_index / num_matrices;

        // Write result
        mat_buffers[mat_buffer_index][mat_buffer_pixel] = array_buffer[array_buffer_index];
    }
}

// Take an array and unflatten it into n same_dimension matrices. Each array's first n elements are the first elements in memory. [arr1_elem1, arr2_elem1, arr3_elem1, arr1_elem2, arr2_elem2, arr3_elem2, ...]
void cuda_unflatten_array_strided(size_t array_id, size_t arr_size, size_t mat_rows, size_t mat_cols, size_t* mat_ids) {
    int mat_size = mat_rows * mat_cols;
    int num_matrices = arr_size / mat_size;

    // Create the buffers for the matrices
    register_matrix_group(mat_rows, mat_cols, num_matrices, mat_ids);
    float** pinned_mat_buffers_ptr = (float**)memory_manager_get_pinned_allocation(sizeof(float*) * num_matrices);
    for (int i = 0; i < num_matrices; i++) {
        pinned_mat_buffers_ptr[i] = get_matrix_gpu_address(mat_ids[i]);
    }

    auto kernel_arg_device_pointers = get_device_kernel_args_pointers(1);
    float** gpu_mat_buffers_dp = (float**)kernel_arg_device_pointers[0];

    // Upload the pointers to a gpu array
    memory_manager_upload_from_pinned_buffer(gpu_mat_buffers_dp, pinned_mat_buffers_ptr, sizeof(float*) * num_matrices);

    // Get the flattened array
    float* gpu_array_buffer = get_matrix_gpu_address(array_id);

    // Kernel launch parameters
    dim3 block_dim(256, 1, 1);
    dim3 grid_dim((arr_size + block_dim.x - 1) / block_dim.x, 1, 1);

    // Run the kernels
    cuda_unflatten_array_strided_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_array_buffer, arr_size, num_matrices, mat_rows, mat_cols, gpu_mat_buffers_dp);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void cuda_center_pad_kernel(float* mat_buffer, int mat_rows, int mat_cols, int pad_rows, int pad_cols, float* out_buffer, int out_rows, int out_cols) {
    const int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    const int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = I[i - pad_rows][j - pad_cols] if in bounds, else 0
        const int input_row = tidY - pad_rows;
        const int input_col = tidX - pad_cols;
        const bool input_row_in_bounds = input_row >= 0 && input_row < mat_rows;
        const bool input_col_in_bounds = input_col >= 0 && input_col < mat_cols;
        const bool in_bounds = input_row_in_bounds && input_col_in_bounds;

        const float result = in_bounds ? mat_buffer[input_row * mat_cols + input_col] : 0.0;
        out_buffer[tidY * out_cols + tidX] = result;
    }
}

size_t cuda_center_pad(size_t mat_id, size_t mat_rows, size_t mat_cols, size_t pad_rows, size_t pad_cols) {
    // Create output buffer
    int out_rows = mat_rows + 2 * pad_rows;
    int out_cols = mat_cols + 2 * pad_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat_buffer = get_matrix_gpu_address(mat_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    cuda_center_pad_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffer, mat_rows, mat_cols, pad_rows, pad_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void cuda_softmax_kernel(float* mat_buffer, int mat_rows, int mat_cols, float* out_buffer) {
    const int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (col < mat_cols) {
        // Go down all the rows, find the max
        float max = -INFINITY;
        for (int row = 0; row < mat_rows; row++) {
            const float val = mat_buffer[row * mat_cols + col];
            max = val > max ? val : max;
        }

        // Now go down all the rows and subtract the max, then exponentiate
        float sum = 0.0;
        for (int row = mat_rows - 1; row >= 0; row--) {
            const float val = mat_buffer[row * mat_cols + col];
            const float exp_val = __expf(val - max);
            out_buffer[row * mat_cols + col] = exp_val;
            sum += exp_val;
        }

        // Now go down all the rows and divide by the sum
        for (int row = 0; row < mat_rows; row++) {
            out_buffer[row * mat_cols + col] /= sum;
        }
    }
}

size_t cuda_softmax(size_t mat_id, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    size_t out_mat_id = register_matrix(mat_rows, mat_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat_buffer = get_matrix_gpu_address(mat_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters, each thread handles one column
    const int THREADS_PER_BLOCK = 128;
    dim3 block_dim(THREADS_PER_BLOCK, 1, 1);
    dim3 grid_dim((mat_cols + block_dim.x - 1) / block_dim.x, 1, 1);

    // Run the kernels
    cuda_softmax_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffer, mat_rows, mat_cols, gpu_out_buffer);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void cuda_crop_kernel(float* mat_buffer, int mat_rows, int mat_cols, int crop_offset_rows, int crop_offset_cols, int crop_rows, int crop_cols, float* out_buffer) {
    const int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    const int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < crop_cols && tidY < crop_rows) {
        // O[i][j] = I[i + crop_offset_rows][j + crop_offset_cols]
        const int input_row = tidY + crop_offset_rows;
        const int input_col = tidX + crop_offset_cols;

        const float result = mat_buffer[input_row * mat_cols + input_col];
        out_buffer[tidY * crop_cols + tidX] = result;
    }
}

size_t cuda_crop(size_t mat_id, size_t mat_rows, size_t mat_cols, size_t crop_offset_rows, size_t crop_offset_cols, size_t crop_rows, size_t crop_cols) {
    // Create output buffer
    size_t out_mat_id = register_matrix(crop_rows, crop_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat_buffer = get_matrix_gpu_address(mat_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters, each thread handles one column
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((crop_cols + block_dim.x - 1) / block_dim.x, (crop_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    cuda_crop_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffer, mat_rows, mat_cols, crop_offset_rows, crop_offset_cols, crop_rows, crop_cols, gpu_out_buffer);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

size_t cuda_copy(size_t mat_id, size_t mat_rows, size_t mat_cols) {
    // We will just use the crop function
    return cuda_crop(mat_id, mat_rows, mat_cols, 0, 0, mat_rows, mat_cols);
}

template <const int block_size>
__global__ void cuda_sum_all_matrix_elements_kernel(float* mat_buffer, int elements_to_sum) {
    // Shared memory for each block. each block handles blockDim elements
    __shared__ float sdata[block_size];

    // Load element into shared
    const int input_index = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[threadIdx.x] = input_index < elements_to_sum ? mat_buffer[input_index] : 0.0;
    __syncthreads();

    // Do reduction in shared memory
    const int sdata_index = threadIdx.x;
    int active_threads = blockDim.x / 2;
    while (active_threads > 0 && sdata_index < active_threads) {
        sdata[sdata_index] += sdata[sdata_index + active_threads];
        __syncthreads();
        active_threads /= 2;
    }

    // Write result for this block to global memory
    if (sdata_index == 0) {
        mat_buffer[blockIdx.x] = sdata[0];
    }
}

size_t cuda_sum_all_matrix_elements(size_t mat_id, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    size_t mat_copy_id = cuda_copy(mat_id, mat_rows, mat_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat_buffer = get_matrix_gpu_address(mat_copy_id);

    // Kernel launch parameters, each thread handles one column
    const int THREADS_PER_BLOCK = 256;
    dim3 block_dim(THREADS_PER_BLOCK, 1, 1);

    // Run the kernels
    int elements_to_sum = mat_rows * mat_cols;
    while (elements_to_sum > 1) {
        dim3 grid_dim((elements_to_sum + block_dim.x - 1) / block_dim.x, 1, 1);
        cuda_sum_all_matrix_elements_kernel<THREADS_PER_BLOCK><<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffer, elements_to_sum);
        gpuErrchk(hipPeekAtLastError());

        elements_to_sum = grid_dim.x;
    }

    // Return just the first element
    size_t result_id = cuda_crop(mat_copy_id, mat_rows, mat_cols, 0, 0, 1, 1);

    // Free the copy
    unregister_matrix(mat_copy_id);

    return result_id;
}

__global__ void cuda_max_by_column_kernel(float* mat_buffer, int mat_rows, int mat_cols, float* out_buffer) {
    const int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (col < mat_cols) {
        // Go down all the rows, find the max
        float max = -INFINITY;
        for (int row = 0; row < mat_rows; row++) {
            const float val = mat_buffer[row * mat_cols + col];
            max = val > max ? val : max;
        }

        // Write result
        out_buffer[col] = max;
    }
}
size_t cuda_max_by_column(size_t mat_id, size_t mat_rows, size_t mat_cols) {
    size_t out_mat_id = register_matrix(1, mat_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat_buffer = get_matrix_gpu_address(mat_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters, each thread handles one column
    const int THREADS_PER_BLOCK = 256;
    dim3 block_dim(THREADS_PER_BLOCK, 1, 1);
    dim3 grid_dim((mat_cols + block_dim.x - 1) / block_dim.x, 1, 1);

    // Run the kernels
    cuda_max_by_column_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffer, mat_rows, mat_cols, gpu_out_buffer);

    return out_mat_id;
}

__global__ void cuda_max_by_row_kernel(float* mat_buffer, int mat_rows, int mat_cols, float* out_buffer) {
    const int row = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < mat_rows) {
        // Go down all the rows, find the max
        float max = -INFINITY;
        for (int col = 0; col < mat_cols; col++) {
            const float val = mat_buffer[row * mat_cols + col];
            max = val > max ? val : max;
        }

        // Write result
        out_buffer[row] = max;
    }
}
size_t cuda_max_by_row(size_t mat_id, size_t mat_rows, size_t mat_cols) {
    size_t out_mat_id = register_matrix(mat_rows, 1);

    // Get the gpu buffers to operate on
    float* gpu_mat_buffer = get_matrix_gpu_address(mat_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters, each thread handles one column
    const int THREADS_PER_BLOCK = 256;
    dim3 block_dim(THREADS_PER_BLOCK, 1, 1);
    dim3 grid_dim((mat_rows + block_dim.x - 1) / block_dim.x, 1, 1);

    // Run the kernels
    cuda_max_by_row_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffer, mat_rows, mat_cols, gpu_out_buffer);

    return out_mat_id;
}

__global__ void cuda_argmax_by_column_kernel(float* mat_buffer, int mat_rows, int mat_cols, float* out_buffer) {
    const int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (col < mat_cols) {
        // Go down all the rows, find the max
        float max = -INFINITY;
        float max_index = 0;
        for (int row = 0; row < mat_rows; row++) {
            const float val = mat_buffer[row * mat_cols + col];
            max_index = val > max ? row : max_index;
            max = val > max ? val : max;
        }

        // Write result
        out_buffer[col] = max_index;
    }
}
size_t cuda_argmax_by_column(size_t mat_id, size_t mat_rows, size_t mat_cols) {
    size_t out_mat_id = register_matrix(1, mat_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat_buffer = get_matrix_gpu_address(mat_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters, each thread handles one column
    const int THREADS_PER_BLOCK = 256;
    dim3 block_dim(THREADS_PER_BLOCK, 1, 1);
    dim3 grid_dim((mat_cols + block_dim.x - 1) / block_dim.x, 1, 1);

    // Run the kernels
    cuda_argmax_by_column_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffer, mat_rows, mat_cols, gpu_out_buffer);

    return out_mat_id;
}

__global__ void cuda_argmax_by_row_kernel(float* mat_buffer, int mat_rows, int mat_cols, float* out_buffer) {
    const int row = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < mat_rows) {
        // Go down all the rows, find the max
        float max = -INFINITY;
        float max_index = 0;
        for (int col = 0; col < mat_cols; col++) {
            const float val = mat_buffer[row * mat_cols + col];
            max_index = val > max ? col : max_index;
            max = val > max ? val : max;
        }

        // Write result
        out_buffer[row] = max_index;
    }
}
size_t cuda_argmax_by_row(size_t mat_id, size_t mat_rows, size_t mat_cols) {
    size_t out_mat_id = register_matrix(mat_rows, 1);

    // Get the gpu buffers to operate on
    float* gpu_mat_buffer = get_matrix_gpu_address(mat_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters, each thread handles one column
    const int THREADS_PER_BLOCK = 256;
    dim3 block_dim(THREADS_PER_BLOCK, 1, 1);
    dim3 grid_dim((mat_rows + block_dim.x - 1) / block_dim.x, 1, 1);

    // Run the kernels
    cuda_argmax_by_row_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffer, mat_rows, mat_cols, gpu_out_buffer);

    return out_mat_id;
}

__global__ void cuda_one_hot_encode_kernel(float* data_buffer, int data_size, int num_classes, float* out_buffer) {
    const int tidX = blockDim.x * blockIdx.x + threadIdx.x;

    if (tidX < data_size) {
        // Each thread handles one row
        // O[i][j] = 1 if j == data[i], else 0
        const int column_to_write_to = data_buffer[tidX];
        out_buffer[tidX * num_classes + column_to_write_to] = 1.0;
    }
}
size_t cuda_one_hot_encode(float* data, size_t data_size, size_t num_classes) {
    // Create output buffer
    size_t out_mat_id = register_matrix(data_size, num_classes);

    // Used pinned transfer for data
    float* pinned_buffer = (float*)memory_manager_get_pinned_allocation(sizeof(float) * data_size);
    memcpy(pinned_buffer, data, sizeof(float) * data_size);

    // Upload the data
    float* gpu_data_buffer = (float*)get_device_kernel_args_pointers(1)[0];
    memory_manager_upload_from_pinned_buffer(gpu_data_buffer, pinned_buffer, sizeof(float) * data_size);

    // Get buffers
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Set the output buffer to 0
    hipMemsetAsync(gpu_out_buffer, 0, sizeof(float) * data_size * num_classes, get_stream());

    // Kernel launch parameters, each thread handles one column
    const int THREADS_PER_BLOCK = 256;
    dim3 block_dim(THREADS_PER_BLOCK, 1, 1);
    dim3 grid_dim((data_size + block_dim.x - 1) / block_dim.x, 1, 1);

    // Run the kernels
    cuda_one_hot_encode_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_data_buffer, data_size, num_classes, gpu_out_buffer);

    return out_mat_id;
}

size_t cuda_one_hot_encode_vector(size_t mat_id, size_t mat_len, size_t num_classes) {
    // Create output buffer
    size_t out_mat_id = register_matrix(mat_len, num_classes);

    // Get buffers
    float* gpu_data_buffer = get_matrix_gpu_address(mat_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Set the output buffer to 0
    hipMemsetAsync(gpu_out_buffer, 0, sizeof(float) * mat_len * num_classes, get_stream());

    // Kernel launch parameters, each thread handles one column
    const int THREADS_PER_BLOCK = 256;
    dim3 block_dim(THREADS_PER_BLOCK, 1, 1);
    dim3 grid_dim((mat_len + block_dim.x - 1) / block_dim.x, 1, 1);

    // Run the kernels
    cuda_one_hot_encode_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_data_buffer, mat_len, num_classes, gpu_out_buffer);

    return out_mat_id;
}

__global__ void element_ln_kernel(float* mat_buffer, int mat_rows, int mat_cols, float* out_buffer) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < mat_cols && tidY < mat_rows) {
        // O[i][j] = ln(mat1[i][j])

        int index = tidY * mat_cols + tidX;
        const float val = mat_buffer[index];
        out_buffer[index] = val <= 0.0 ? 0.0 : log(mat_buffer[index]);  // Handle <= 0 to avoid NAN
    }
}

size_t cuda_element_ln(size_t mat_id, size_t mat_rows, size_t mat_cols, bool inplace) {
    // Create output buffer
    size_t out_rows = mat_rows;
    size_t out_cols = mat_cols;
    size_t out_mat_id = inplace ? mat_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat_buffer = get_matrix_gpu_address(mat_id);
    float* gpu_out_buffer = get_matrix_gpu_address(out_mat_id);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols + block_dim.x - 1) / block_dim.x, (out_rows + block_dim.y - 1) / block_dim.y, 1);

    // Run the kernels
    element_ln_kernel<<<grid_dim, block_dim, 0, get_stream()>>>(gpu_mat_buffer, mat_rows, mat_cols, gpu_out_buffer);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}
