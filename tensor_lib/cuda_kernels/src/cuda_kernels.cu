#include "hip/hip_runtime.h"
#include <unordered_map>
#include <vector>

#include "./cuda_kernels.cuh"

bool init_cublas = false;
bool init_pool = false;
hipblasHandle_t handle;
size_t mat_generated_count(0);
std::unordered_map<size_t, float*> mat_map;

// Error checking macro: https://stackoverflow.com/a/14038590
#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = false) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/////////////////////
/// TEST FUNCTIONS
/////////////////////
__global__ void test_kernel() {
    printf("Hello from the kernel!\n");
}

__global__ void test_kernel_2(int* result) {
    *result = 8;
}

void test() {
    int result;
    int* d_result;
    test_kernel<<<1, 1>>>();
    gpuErrchk(hipPeekAtLastError());
    hipDeviceSynchronize();
    if (hipPeekAtLastError() != hipSuccess) {
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
    }

    hipError_t err = hipMalloc((void**)&d_result, sizeof(int));
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }

    test_kernel_2<<<1, 1>>>(d_result);
    if (hipPeekAtLastError() != hipSuccess) {
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
    }

    err = hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    hipFree(d_result);

    printf("Result: %d\n", result);
    printf("Finished Running Kernels.");
}

void test_array_fill(float* buffer, size_t length) {
    for (size_t i = 0; i < length; i++) {
        buffer[i] = i;
    }
}

void cuda_synchronize() {
    hipDeviceSynchronize();
}

/////////////////////
/// Matrix Setup API
/////////////////////
void init_cublas_handle() {
    hipblasCreate(&handle);
    hipblasSetStream(handle, 0);
    init_cublas = true;
}
void init_min_pool_size() {
    int device;
    hipGetDevice(&device);
    hipMemPool_t mempool;
    hipDeviceGetDefaultMemPool(&mempool, device);
    size_t threshold = sizeof(float) * 2048 * 2048;  // Around 68 Mb reserved
    hipMemPoolSetAttribute(mempool, hipMemPoolAttrReleaseThreshold, &threshold);
    init_pool = false;
}
size_t register_matrix_buffer(float* gpu_buffer) {
    if (init_pool) {
        init_min_pool_size();
    }

    // Register with the map for retrieval later
    mat_map[mat_generated_count] = gpu_buffer;
    return mat_generated_count++;  // Fine if this overflows
}

size_t register_matrix(size_t rows, size_t cols) {
    // Upload the data
    float* gpu_buffer;
    gpuErrchk(hipMallocAsync(&gpu_buffer, sizeof(float) * rows * cols, 0));

    return register_matrix_buffer(gpu_buffer);
}

size_t register_matrix(float* data, size_t rows, size_t cols) {
    // Upload the data
    float* gpu_buffer;
    gpuErrchk(hipMallocAsync(&gpu_buffer, sizeof(float) * rows * cols, 0));
    gpuErrchk(hipMemcpy(gpu_buffer, data, sizeof(float) * rows * cols, hipMemcpyHostToDevice));
    // Potentially nasty bug by acting like you copied data when you havent finished if using cudaMemCpyAsync...
    return register_matrix_buffer(gpu_buffer);
}

void unregister_matrix(size_t mat_id) {
    gpuErrchk(hipFreeAsync(mat_map[mat_id], 0));
    mat_map.erase(mat_id);
}

void get_matrix_data(size_t mat_id, int rows, int cols, float* data_buffer) {
    float* gpu_buffer = mat_map[mat_id];
    gpuErrchk(hipMemcpy(data_buffer, gpu_buffer, sizeof(float) * rows * cols, hipMemcpyDeviceToHost));
}

//////////////////////////
/// Matrix Operations API
//////////////////////////
/// TODO: Possibly grid stride. Optimize transpose. Optimize matmult.
__global__ void element_add_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;
    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] + mat2[i][j]
        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] + mat2_buffer[index];
    }
}
size_t cuda_element_add(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Create output buffer`
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);
    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];
    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);
    // Run the kernels
    element_add_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());
    // Return result matrix id
    return out_mat_id;
}

__global__ void element_subtract_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;
    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] - mat2[i][j]

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] - mat2_buffer[index];
    }
}

size_t cuda_element_subtract(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    element_subtract_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void element_multiply_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] * mat2[i][j]

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] * mat2_buffer[index];
    }
}

size_t cuda_element_multiply(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    element_multiply_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void scalar_multiply_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float scalar, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] * scalar

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] * scalar;
    }
}

size_t cuda_scalar_multiply(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, float scalar, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    scalar_multiply_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, scalar, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void matrix_multiply_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][:] weighted sum mat2[:][j]
        // Where common dimension : is mat1col/mat2row

        float weighted_sum = 0.0;
        for (int common = 0; common < mat1_cols; common++) {
            // mat1[i][common]
            int mat1_index = mat1_cols * tidX + common;
            // mat1[common][j]
            int mat2_index = mat2_cols * common + tidY;

            weighted_sum += mat1_buffer[mat1_index] * mat2_buffer[mat2_index];
        }

        int output_index = tidX * out_cols + tidY;
        out_buffer[output_index] = weighted_sum;
    }
}

__global__ void matrix_multiply_kernel_2(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    // Go by col row instead of row col. Enabled memory coalescing
    const int col = blockDim.x * blockIdx.x + threadIdx.x;
    const int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (row >= out_rows || col >= out_cols) {
        return;
    }

    // O[i][j] = mat1[i][:] weighted sum mat2[:][j]
    // Where common dimension : is mat1col/mat2row

    float weighted_sum = 0.0;
    for (int common = 0; common < mat1_cols; common++) {
        // mat1[i][common]
        int mat1_index = mat1_cols * row + common;
        // mat1[common][j]
        int mat2_index = mat2_cols * common + col;

        weighted_sum += mat1_buffer[mat1_index] * mat2_buffer[mat2_index];
    }

    const int output_index = row * out_cols + col;
    out_buffer[output_index] = weighted_sum;
}

__global__ void matrix_multiply_kernel_3(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    const int block_dim = 32;
    const int block_area = block_dim * block_dim;

    // Block tiling with shared memory
    __shared__ float s_mat1[block_area];
    __shared__ float s_mat2[block_area];

    const int block_row = blockIdx.y;
    const int block_col = blockIdx.x;

    int mat1_block_pos = block_row * block_dim * mat1_cols;
    int mat2_block_pos = block_col * block_dim;
    int out_block_pos = block_row * block_dim * out_cols + block_col * block_dim;

    // So within our block we are gonna figure out this thread's position
    int thread_row = threadIdx.y;
    int thread_col = threadIdx.x;

    int out_row = block_row * block_dim + thread_row;
    int out_col = block_col * block_dim + thread_col;
    if (out_row >= out_rows || out_col >= out_cols) {
        return;
    }

    float weighted_sum = 0.0;
    int common_partial_block = mat1_cols % block_dim;
    int common_in_block = mat1_cols - common_partial_block;
    for (int k = 0; k < common_in_block; k += block_dim) {
        s_mat1[thread_row * block_dim + thread_col] = mat1_buffer[mat1_block_pos + thread_row * mat1_cols + thread_col];
        s_mat2[thread_row * block_dim + thread_col] = mat2_buffer[mat2_block_pos + thread_row * mat2_cols + thread_col];
        __syncthreads();

        mat1_block_pos += block_dim;
        mat2_block_pos += block_dim * mat2_cols;
        for (int i = 0; i < block_dim; i++) {
            weighted_sum += s_mat1[thread_row * block_dim + i] * s_mat2[i * block_dim + thread_col];
        }
        __syncthreads();
    }

    // Handle partial block case
    s_mat1[thread_row * block_dim + thread_col] = mat1_buffer[mat1_block_pos + thread_row * mat1_cols + thread_col];
    s_mat2[thread_row * block_dim + thread_col] = mat2_buffer[mat2_block_pos + thread_row * mat2_cols + thread_col];
    __syncthreads();

    mat1_block_pos += block_dim;
    mat2_block_pos += block_dim * mat2_cols;
    for (int i = 0; i < common_partial_block; i++) {
        weighted_sum += s_mat1[thread_row * block_dim + i] * s_mat2[i * block_dim + thread_col];
    }

    out_buffer[out_block_pos + (thread_row * out_cols) + thread_col] = weighted_sum;
}

// block_M is rows in mat1 shared block
// block_N is cols in mat2 shared block
// block_k is shared dimensions for shared block. Also the # of results each thread will compute in C
// For this to work we want the shared dimension block_K to be smaller than block_M and block_N
// This way, multiple threads reuse sections from mat1 and mat2 ,with more output work
// Example: bK is 8 while bM and bN are 64. Output is a 64x64 area.
//          So you can spin up 512 threads per block. They load vram->shared
//          Then each thread can work on 8 pieces of the output 64x64 area (64*64/8 = 512)
template <const int block_M, const int block_N, const int block_K>
__global__ void matrix_multiply_kernel_4(int M, int N, int K, float* mat1_buffer, float* mat2_buffer, float* out_buffer) {
    // Block tiling with shared memory
    // Each one of these threads will handle #block_K output result columns
    __shared__ float s_mat1[block_M * block_K];
    __shared__ float s_mat2[block_K * block_N];

    float thread_results[block_K] = {0.0};

    const int block_row = blockIdx.y;
    const int block_col = blockIdx.x;

    // Get starting positions of each block
    int mat1_block_pos = block_row * block_M * K;
    int mat2_block_pos = block_col * block_N;
    int out_block_pos = block_row * block_M * N + block_col * block_N;

    // Used to track if out of bounds
    const int mat1_load_index_row = block_row * block_M + threadIdx.x;
    const int mat2_load_index_col = block_col * block_N + threadIdx.x;
    int mat_common_index = threadIdx.y;
    const bool exceeded_mat1_row = mat1_load_index_row >= M;
    const bool exceeded_mat2_col = mat2_load_index_col >= N;

    // outer loop over block tiles
    for (unsigned int common_block = 0; common_block < K; common_block += block_K) {
        const int within_mat1 = (int)!(exceeded_mat1_row || mat_common_index >= K);
        const int within_mat2 = (int)!(mat_common_index >= K || exceeded_mat2_col);
        int mat1_load_index = mat1_block_pos + threadIdx.x * K + threadIdx.y;
        int mat2_load_index = mat2_block_pos + threadIdx.y * N + threadIdx.x;

        // Prevent loading OOB
        mat1_load_index *= within_mat1;
        mat2_load_index *= within_mat2;

        // Load block data into shared memory. Load 0 is OOB.
        s_mat1[threadIdx.x * block_K + threadIdx.y] = mat1_buffer[mat1_load_index] * within_mat1;
        s_mat2[threadIdx.y * block_N + threadIdx.x] = mat2_buffer[mat2_load_index] * within_mat2;
        __syncthreads();

        // Advance block
        mat1_block_pos += block_K;
        mat2_block_pos += block_K * N;
        mat_common_index += block_K;

        // Go through common dimensions of block (across row of mat1 and down col of mat2)
        for (unsigned int block_common_index = 0; block_common_index < block_K; ++block_common_index) {
            const float shared_mat2_val = s_mat2[block_common_index * block_N + threadIdx.x];

            // Now this thread will accumulate the result for each t_row in the t_col of C
            for (unsigned int result_index = 0; result_index < block_K; ++result_index) {
                thread_results[result_index] +=
                    s_mat1[(threadIdx.y * block_K + result_index) * block_K + block_common_index] * shared_mat2_val;
            }
        }
        __syncthreads();
    }

    // Write results with bounds checking
    const int out_index_row = block_row * block_M + threadIdx.y * block_K;
    const int out_index_col = block_col * block_N + threadIdx.x;

    for (int i = 0; i < block_K; i++) {
        if (out_index_row + i < M && out_index_col < N) {
            out_buffer[out_block_pos + (threadIdx.y * block_K + i) * N + threadIdx.x] = thread_results[i];
        }
    }
}

// block_M is rows in mat1 shared block
// block_N is cols in mat2 shared block
// block_k is shared dimensions for shared block.
// The thread will calculate block_k * block_k results (So now a 2d version of v3)
// For this to work we want the shared dimension block_K to be extremely smaller than block_M and block_N
// This way, multiple threads reuse sections from mat1 and mat2 ,with more output work
// Example: bK is 8 while bM and bN are 128. Output is a 128x128 area.
//          So you can spin up 256 threads per block. They load vram->shared
//          Then each thread can work on 8x8 pieces of the output 128x128 area (128x128/64 = 256)
//          You might be wondering why not 512 threads like previously?
//          Well that increases the mem requirements per block, reducing occupancy.
template <const int block_M, const int block_N, const int block_K>
__global__ void matrix_multiply_kernel_5(int M, int N, int K, float* __restrict__ mat1_buffer, float* __restrict__ mat2_buffer, float* __restrict__ out_buffer) {
    // 2D Block tiling with shared memory
    __shared__ float s_mat1[block_M * block_K];
    __shared__ float s_mat2[block_K * block_N];

    float thread_results[block_K * block_K] = {0.0};

    const int block_row = blockIdx.y;
    const int block_col = blockIdx.x;

    // Output within block details
    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int out_block_row = tid / (block_M / block_K);
    const int out_block_col = tid % (block_N / block_K);

    const int num_threads_per_block = blockDim.x * blockDim.y;
    const int num_elements_to_load = (block_M * block_K) / num_threads_per_block;

    const int stride_mat1 = num_threads_per_block / block_K;
    const int stride_mat2 = num_threads_per_block / block_N;

    int mat1_pos = block_row * block_M * K;
    int mat2_pos = block_col * block_N;

// outer loop over block tiles
#pragma unroll
    for (int common_block = 0; common_block < K; common_block += block_K) {
#pragma unroll 4
        for (int i = 0; i < num_elements_to_load; i++) {
            const int mat1_row_within_block = (threadIdx.x + stride_mat1 * i);
            const int mat1_col_within_block = threadIdx.y;
            const int mat2_row_within_block = (threadIdx.y / num_elements_to_load) + i * stride_mat2;
            const int mat2_col_within_block = (threadIdx.y % num_elements_to_load) * blockDim.x + threadIdx.x;

            const int mat1_load_index_row = block_row * block_M + mat1_row_within_block;
            const int mat1_load_index_col = common_block + mat1_col_within_block;
            const int mat2_load_index_row = common_block + mat2_row_within_block;
            const int mat2_load_index_col = block_col * block_N + mat2_col_within_block;

            const bool exceeded_mat1_row = mat1_load_index_row >= M;
            const bool exceeded_mat1_col = mat1_load_index_col >= K;
            const bool exceeded_mat2_row = mat2_load_index_row >= K;
            const bool exceeded_mat2_col = mat2_load_index_col >= N;

            const int within_mat1 = (int)!(exceeded_mat1_row || exceeded_mat1_col);
            const int within_mat2 = (int)!(exceeded_mat2_row || exceeded_mat2_col);
            int mat1_load_index = mat1_pos + mat1_row_within_block * K + mat1_col_within_block;
            int mat2_load_index = mat2_pos + mat2_row_within_block * N + mat2_col_within_block;

            mat1_load_index *= within_mat1;
            mat2_load_index *= within_mat2;

            s_mat1[mat1_row_within_block * block_K + mat1_col_within_block] =
                mat1_buffer[mat1_load_index] * within_mat1;
            s_mat2[mat2_row_within_block * block_N + mat2_col_within_block] =
                mat2_buffer[mat2_load_index] * within_mat2;
        }

        mat1_pos += block_K;
        mat2_pos += block_K * N;

        __syncthreads();

        // Go through common dimensions of block (across row of mat1 and down col of mat2)
#pragma unroll 8
        for (int block_common_index = 0; block_common_index < block_K; block_common_index++) {
            // Now this thread will accumulate the block_K x block_K results from shared memory
#pragma unroll 8
            for (int result_index_row = 0; result_index_row < block_K; result_index_row++) {
#pragma unroll 8
                for (int result_index_col = 0; result_index_col < block_K; result_index_col++) {
                    thread_results[result_index_row * block_K + result_index_col] +=
                        s_mat1[(out_block_row * block_K + result_index_row) * block_K + block_common_index] *
                        s_mat2[(block_common_index * block_N) + (out_block_col * block_K + result_index_col)];
                }
            }
        }
        __syncthreads();
    }

    // Write results with bounds checking
    const int out_index_row = block_row * block_M + out_block_row * block_K;
    const int out_index_col = block_col * block_N + out_block_col * block_K;

#pragma unroll 8
    for (int i = 0; i < block_K; i++) {
#pragma unroll 8
        for (int j = 0; j < block_K; j++) {
            if (out_index_row + i < M && out_index_col + j < N) {
                out_buffer[(out_index_row + i) * N + out_index_col + j] = thread_results[i * block_K + j];
            }
        }
    }
}

size_t cuda_matrix_multiply(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat2_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    // const int THREADS_PER_BLOCK_X = 32;
    // const int THREADS_PER_BLOCK_Y = 32;

    // dim3 block_dim(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1);
    // dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // // Run the kernels
    // matrix_multiply_kernel_3<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);

    // V4 launch
    const int M = mat1_rows;
    const int N = mat2_cols;
    const int K = mat1_cols;

    const int THREADS_PER_BLOCK_X = 32;
    const int THREADS_PER_BLOCK_Y = 8;

    dim3 block_dim(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
    dim3 grid_dim((N / 128) + 1, (M / 128) + 1, 1);
    matrix_multiply_kernel_5<128, 128, 8><<<grid_dim, block_dim>>>(M, N, K, gpu_mat1_buffer, gpu_mat2_buffer, gpu_out_buffer);

    // CUBLAS version (for comparison to mine)
    // if (!init_cublas) {
    //     init_cublas_handle();
    // }
    // float alpha = 1.0;
    // float beta = 0.0;
    // hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, mat2_cols, mat1_rows, mat1_cols, &alpha, gpu_mat2_buffer, mat2_cols, gpu_mat1_buffer, mat1_cols, &beta, gpu_out_buffer, mat2_cols);

    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void add_vector_to_columns_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] + mat2[i][0]

        int mat1_index = tidY * mat1_cols + tidX;
        int mat2_index = tidY;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] + mat2_buffer[mat2_index];
    }
}

__global__ void add_vector_to_rows_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] + mat2[0][j]

        int mat1_index = tidY * mat1_cols + tidX;
        int mat2_index = tidX;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] + mat2_buffer[mat2_index];
    }
}

size_t cuda_add_vector(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Determine orientation
    bool is_column_vector = (mat2_cols == 1 && mat2_rows == mat1_rows);

    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    if (is_column_vector) {
        add_vector_to_columns_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    } else {
        add_vector_to_rows_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    }
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void divide_by_column_vector_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] / mat2[i][0]

        int mat1_index = tidY * mat1_cols + tidX;
        int mat2_index = tidY;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] / mat2_buffer[mat2_index];
    }
}

__global__ void divide_by_row_vector_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] / mat2[0][j]

        int mat1_index = tidY * mat1_cols + tidX;
        int mat2_index = tidX;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] / mat2_buffer[mat2_index];
    }
}

size_t cuda_divide_by_vector(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Determine orientation
    bool is_column_vector = (mat2_cols == 1 && mat2_rows == mat1_rows);

    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    if (is_column_vector) {
        divide_by_column_vector_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    } else {
        divide_by_row_vector_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    }
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void element_exp_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = exp(mat1[i][j])

        int index = tidY * out_cols + tidX;
        out_buffer[index] = exp(mat1_buffer[index]);  // Also available __exp for fast
    }
}

size_t cuda_element_exp(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    element_exp_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void element_ReLU_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = x if x>0 else 0

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] > 0 ? mat1_buffer[index] : 0.0;
    }
}

size_t cuda_element_ReLU(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    element_ReLU_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void element_ReLU_prime_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = x if x>0 else 1

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] == 0.0 ? 0.0 : 1.0;
    }
}

size_t cuda_element_ReLU_prime(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    element_ReLU_prime_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void sum_rows_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][0] = sum (mat1[i][:])

        float row_sum = 0.0;
        int mat1_row_start_index = tidY * mat1_cols;
        for (int i = 0; i < mat1_cols; i++) {
            int mat1_index = mat1_row_start_index + i;
            row_sum += mat1_buffer[mat1_index];
        }

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = row_sum;
    }
}

size_t cuda_sum_rows(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = 1;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    sum_rows_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void sum_columns_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[0][j] = sum (mat1[:][j])

        float col_sum = 0.0;
        for (int i = 0; i < mat1_rows; i++) {
            int mat1_index = tidX + i * mat1_cols;
            col_sum += mat1_buffer[mat1_index];
        }

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = col_sum;
    }
}

size_t cuda_sum_columns(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = 1;
    int out_cols = mat1_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    sum_columns_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void transpose_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[j][i]

        int mat1_index = tidX * mat1_cols + tidY;

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = mat1_buffer[mat1_index];
    }
}

size_t cuda_transpose(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = mat1_cols;
    int out_cols = mat1_rows;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    transpose_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void cuda_max_pool_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols, float* gpu_max_bitmask) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // For each 2x2 area pick the maximum value
        // We will mem coalesce by getting first two in row 1
        // Then next 2 in row2

        // Grab data w/t bounds check
        // TODO: Bounds check

        int block_start_row = tidY * 2;
        int block_start_col = tidX * 2;
        int block_start = block_start_row * mat1_cols + block_start_col;

        // bool block_00_oob = false;
        bool block_01_oob = (block_start_col + 1) >= mat1_cols;
        bool block_10_oob = (block_start_row + 1) >= mat1_rows;
        bool block_11_oob = block_01_oob || block_10_oob;

        float small_float = -1e30;  // Should probably use FLT_MIN but language server no like it

        // TODO: Use bit operations instead of ternary (it's faster idk why the compiler can't figure it out)
        float block_00 = mat1_buffer[block_start];
        float block_01 = block_01_oob ? small_float : mat1_buffer[block_start + 1];
        block_start += mat1_cols;
        float block_10 = block_10_oob ? small_float : mat1_buffer[block_start];
        float block_11 = block_11_oob ? small_float : mat1_buffer[block_start + 1];

        float result = max(max(block_00, block_01), max(block_10, block_11));

        if (result == block_00) {
            gpu_max_bitmask[block_start_row * mat1_cols + block_start_col] = 1.0;
        } else if (result == block_01) {
            gpu_max_bitmask[block_start_row * mat1_cols + block_start_col + 1] = 1.0;
        } else if (result == block_10) {
            gpu_max_bitmask[(block_start_row + 1) * mat1_cols + block_start_col] = 1.0;
        } else if (result == block_11) {
            gpu_max_bitmask[(block_start_row + 1) * mat1_cols + block_start_col + 1] = 1.0;
        }

        // Write maxpool result
        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = result;
    }
}

// 2x2 since other reduction sizes are not really used
Tuple cuda_max_pool(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = mat1_rows / 2 + mat1_rows % 2;
    int out_cols = mat1_cols / 2 + mat1_cols % 2;
    size_t out_mat_id = register_matrix(out_rows, out_cols);
    size_t max_bitmask = register_matrix(mat1_rows, mat1_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];
    float* gpu_max_bitmask = mat_map[max_bitmask];

    // Zero out bitmask
    hipMemset(gpu_max_bitmask, 0.0, mat1_rows * mat1_cols * sizeof(float));

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    cuda_max_pool_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols, gpu_max_bitmask);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return Tuple{out_mat_id, max_bitmask};
}

__global__ void cuda_nearest_neighbor_2x_upsample_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    // Upsample by nearest neighbor
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i/2][j/2]
        int mat1_index = (tidY / 2) * mat1_cols + (tidX / 2);

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = mat1_buffer[mat1_index];
    }
}

// Odd upsample will leave out one row and one column from the upsampled matrix
size_t cuda_nearest_neighbor_2x_upsample(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, bool odd_upsample) {
    // Create output buffer
    int out_rows = mat1_rows * 2 - (int)odd_upsample;
    int out_cols = mat1_cols * 2 - (int)odd_upsample;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK_X = 16;
    const int THREADS_PER_BLOCK_Y = 16;

    dim3 block_dim(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);
    cuda_nearest_neighbor_2x_upsample_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void cuda_rotate_180_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // Rotating an array 180 means
        // x_output = length - x_current
        // y_output = height - y_current
        int x_out = mat1_cols - tidX - 1;
        int y_out = mat1_rows - tidY - 1;
        float input = mat1_buffer[tidY * mat1_cols + tidX];

        int output_index = y_out * out_cols + x_out;
        out_buffer[output_index] = input;
    }
}

size_t cuda_rotate_180(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    cuda_rotate_180_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Naive implementation
__global__ void cuda_convolution_kernel_valid_1(float* mat1_buffer, int mat1_rows, int mat1_cols, float* kernel_buffer, int kernel_rows, int kernel_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;
    int threadIdWithinBlock = threadIdx.y * blockDim.x + threadIdx.x;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = weighted sum of kernel with input, where kernel is kept within bounds of input
        float result = 0.0;
        const int kernel_top_left_row = tidY;
        const int kernel_top_left_col = tidX;

#pragma unroll 3
        for (int m = 0; m < kernel_rows; m++) {
#pragma unroll 3
            for (int n = 0; n < kernel_cols; n++) {
                const float mat1_val = mat1_buffer[(kernel_top_left_row + m) * mat1_cols + (kernel_top_left_col + n)];
                const float kernel_val = kernel_buffer[m * kernel_cols + n];
                result += mat1_val * kernel_val;
            }
        }

        const int out_index = tidY * out_cols + tidX;
        out_buffer[out_index] = result;
    }
}

// Be careful, this needs to be optimized or your CNN will suffer
size_t cuda_convolution_valid(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t kernel_id, size_t kernel_rows, size_t kernel_cols) {
    // Create output buffer
    // Dimension of output is input - kernel + 1
    int out_rows = mat1_rows - kernel_rows + 1;
    int out_cols = mat1_cols - kernel_cols + 1;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_kernel_buffer = mat_map[kernel_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    cuda_convolution_kernel_valid_1<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_kernel_buffer, kernel_rows, kernel_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Naive implementation
__global__ void cuda_convolution_kernel_same_1(float* mat1_buffer, int mat1_rows, int mat1_cols, float* kernel_buffer, int kernel_rows, int kernel_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = weighted sum of kernel with input, where kernel is centered at i,j

        float result = 0.0;
        const int apothem = kernel_rows / 2;
#pragma unroll 3
        for (int m = 0; m < kernel_rows; m++) {
#pragma unroll 3
            for (int n = 0; n < kernel_cols; n++) {
                int input_row = m - apothem + tidY;
                int input_col = n - apothem + tidX;
                bool input_row_in_bounds = input_row >= 0 && input_row < mat1_rows;
                bool input_col_in_bounds = input_col >= 0 && input_col < mat1_cols;

                if (input_row_in_bounds && input_col_in_bounds) {
                    const int curr_mat1_index = input_row * mat1_cols + input_col;
                    const int curr_kernel_index = m * kernel_cols + n;
                    result += mat1_buffer[curr_mat1_index] * kernel_buffer[curr_kernel_index];
                }
            }
        }

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = result;
    }
}

// Convolution is zero-padded (Output is the same size as input)
// Expects odd size, square kernels ONLY
// Be careful, this needs to be optimized or your CNN will suffer
size_t cuda_convolution_same(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t kernel_id, size_t kernel_rows, size_t kernel_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_kernel_buffer = mat_map[kernel_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    cuda_convolution_kernel_same_1<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_kernel_buffer, kernel_rows, kernel_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

// Naive implementation
__global__ void cuda_convolution_kernel_full_1(float* mat1_buffer, int mat1_rows, int mat1_cols, float* kernel_buffer, int kernel_rows, int kernel_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = weighted sum of kernel with input, where kernel is centered at i,j
        float result = 0.0;
        const int input_start_row = (-kernel_rows + 1) + tidY;
        const int input_start_col = (-kernel_cols + 1) + tidX;
        for (int m = 0; m < kernel_rows; m++) {
            for (int n = 0; n < kernel_cols; n++) {
                int input_row = input_start_row + m;
                int input_col = input_start_col + n;
                bool input_row_in_bounds = input_row >= 0 && input_row < mat1_rows;
                bool input_col_in_bounds = input_col >= 0 && input_col < mat1_cols;

                if (input_row_in_bounds && input_col_in_bounds) {
                    const int curr_mat1_index = input_row * mat1_cols + input_col;
                    const int curr_kernel_index = m * kernel_cols + n;
                    result += mat1_buffer[curr_mat1_index] * kernel_buffer[curr_kernel_index];
                }
            }
        }

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = result;
    }
}

// Be careful, this needs to be optimized or your CNN will suffer
size_t cuda_convolution_full(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t kernel_id, size_t kernel_rows, size_t kernel_cols) {
    // Create output buffer
    // Dimension of output is input + kernel - 1
    int out_rows = mat1_rows + kernel_rows - 1;
    int out_cols = mat1_cols + kernel_cols - 1;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_kernel_buffer = mat_map[kernel_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 16;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    cuda_convolution_kernel_full_1<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_kernel_buffer, kernel_rows, kernel_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

size_t cuda_convolution(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t kernel_id, size_t kernel_rows, size_t kernel_cols, ConvolutionType conv_type) {
    if (conv_type == ConvolutionType::VALID) {
        return cuda_convolution_valid(mat1_id, mat1_rows, mat1_cols, kernel_id, kernel_rows, kernel_cols);
    } else if (conv_type == ConvolutionType::SAME) {
        return cuda_convolution_same(mat1_id, mat1_rows, mat1_cols, kernel_id, kernel_rows, kernel_cols);
    } else if (conv_type == ConvolutionType::FULL) {
        return cuda_convolution_full(mat1_id, mat1_rows, mat1_cols, kernel_id, kernel_rows, kernel_cols);
    }
}

__global__ void cuda_img2col_valid(float** mat_buffers, int input_depth, int input_rows, int input_cols, int filter_depth, int filter_rows, int filter_cols, float* out_buffer, int out_rows, int out_cols) {
    const int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    const int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    // This thread will handle one patch of the image, through all the kernels
    // This means each thread handle one column of the output
    const int number_of_patches = out_cols;
    const int current_patch = tidX;

    if (current_patch < number_of_patches) {
// Go through each of the kernels
#pragma unroll
        for (int curr_channel = 0; curr_channel < input_depth; curr_channel++) {
            const float* current_buffer = mat_buffers[curr_channel];

            // Now translate current_patch into the patch's top left corner
            const int kernel_top_left_row = current_patch / (input_cols - filter_cols + 1);
            const int kernel_top_left_col = current_patch % (input_cols - filter_cols + 1);
            const int base_output_row = curr_channel * filter_rows * filter_cols;
            const int output_col = current_patch;

// Now construct the patch
#pragma unroll 3
            for (int m = 0; m < filter_rows; m++) {
#pragma unroll 3
                for (int n = 0; n < filter_cols; n++) {
                    const float mat_val = current_buffer[(kernel_top_left_row + m) * input_cols + (kernel_top_left_col + n)];
                    const int output_index = base_output_row + m * filter_cols + n;
                    out_buffer[output_index * out_cols + output_col] = mat_val;
                }
            }
        }
    }
}

size_t cuda_img2col_valid(size_t* mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, size_t kernel_rows, size_t kernel_cols) {
    // Create output buffer
    const int kernel_count = num_matrices;  // num_matrices is the input depth
    const int out_rows = kernel_count * kernel_rows * kernel_cols;
    const int out_cols = (mat_rows - kernel_rows + 1) * (mat_cols - kernel_cols + 1);
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    std::vector<float*> mat_buffers;
    for (size_t i = 0; i < num_matrices; i++) {
        mat_buffers.push_back(mat_map[mat_ids[i]]);
    }

    float** gpu_mat_buffers;
    hipMallocAsync(&gpu_mat_buffers, sizeof(float*) * num_matrices, 0);
    hipMemcpy(gpu_mat_buffers, &mat_buffers[0], sizeof(float*) * num_matrices, hipMemcpyHostToDevice);
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    // Let's have each threads handle its own patch between all the kernels
    // So we will calculate the number of patches == number of columns
    // Data access should be coalesced this way
    const int THREADS_PER_BLOCK = 1024;
    dim3 block_dim(THREADS_PER_BLOCK, 1, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, 1, 1);

    // Run the kernels
    cuda_img2col_valid<<<grid_dim, block_dim>>>(gpu_mat_buffers, num_matrices, mat_rows, mat_cols, kernel_count, kernel_rows, kernel_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Cleanup
    hipFreeAsync((void*)gpu_mat_buffers, 0);

    // Return result matrix id
    return out_mat_id;
}

// Take an image and convert it to a matrix of columns based on patches (with specified padding) the filter makes of image
size_t cuda_img2col(size_t* mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols, size_t kernel_rows, size_t kernel_cols, ConvolutionType conv_type) {
    if (conv_type == ConvolutionType::VALID) {
        return cuda_img2col_valid(mat_ids, num_matrices, mat_rows, mat_cols, kernel_rows, kernel_cols);
    } else if (conv_type == ConvolutionType::SAME) {
        return 0;
    } else if (conv_type == ConvolutionType::FULL) {
        return 0;
    }
}

__global__ void cuda_flatten_array_kernel(float** mat_buffers, int mat_rows, int mat_cols, float* out_buffer, int out_rows, int out_cols) {
    const int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    const int tidY = blockDim.y * blockIdx.y + threadIdx.y;
    const int output_index = tidY * out_cols + tidX;
    const int output_img_size = out_rows * out_cols;
    const int each_input_img_size = mat_rows * mat_cols;

    if (output_index < output_img_size) {
        // Grab the gpu buffer we are reffering to
        const int current_buffer_index = output_index / each_input_img_size;
        const float* current_buffer = mat_buffers[current_buffer_index];

        // Determine the pixel to copy
        const int current_buffer_pixel = output_index % each_input_img_size;

        // Write result
        out_buffer[output_index] = current_buffer[current_buffer_pixel];
    }
}

// Take n same_dimension matrices and flatten them into an array
size_t cuda_flatten_array(size_t* mat_ids, size_t num_matrices, size_t mat_rows, size_t mat_cols) {
    // Create output buffer
    int out_rows = 1;
    int out_cols = num_matrices * mat_rows * mat_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    std::vector<float*> mat_buffers;
    for (size_t i = 0; i < num_matrices; i++) {
        mat_buffers.push_back(mat_map[mat_ids[i]]);
    }

    float** gpu_mat_buffers;
    hipMallocAsync(&gpu_mat_buffers, sizeof(float*) * num_matrices, 0);
    hipMemcpy(gpu_mat_buffers, &mat_buffers[0], sizeof(float*) * num_matrices, hipMemcpyHostToDevice);
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    dim3 block_dim(256, 1, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    cuda_flatten_array_kernel<<<grid_dim, block_dim>>>(gpu_mat_buffers, mat_rows, mat_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Cleanup
    hipFreeAsync((void*)gpu_mat_buffers, 0);

    // Return result matrix id
    return out_mat_id;
}

__global__ void cuda_unflatten_array_kernel(float* array_buffer, int arr_size, int mat_rows, int mat_cols, float** mat_buffers) {
    const int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    const int array_buffer_index = tidX;

    if (array_buffer_index < arr_size) {
        // Check which mat_buffer to write to
        const int mat_size = mat_rows * mat_cols;
        const int mat_buffer_index = array_buffer_index / mat_size;
        const int mat_buffer_pixel = array_buffer_index % mat_size;

        // Write result
        mat_buffers[mat_buffer_index][mat_buffer_pixel] = array_buffer[array_buffer_index];
    }
}

// Take an array and unflatten it into n same_dimension matrices.
void cuda_unflatten_array(size_t array_id, size_t arr_size, size_t mat_rows, size_t mat_cols, size_t* mat_ids) {
    int mat_size = mat_rows * mat_cols;
    int num_matrices = arr_size / mat_size;

    // Create the buffers for the matrices
    std::vector<float*> gpu_mat_buffers;
    for (int i = 0; i < num_matrices; i++) {
        size_t mat_id = register_matrix(mat_rows, mat_cols);
        gpu_mat_buffers.push_back(mat_map[mat_id]);

        // Write back to rust vector
        mat_ids[i] = mat_id;
    }

    // Upload the gpu_mat_buffers to the gpu
    float** gpu_mat_buffers_ptr;
    hipMallocAsync(&gpu_mat_buffers_ptr, sizeof(float*) * num_matrices, 0);
    hipMemcpy(gpu_mat_buffers_ptr, &gpu_mat_buffers[0], sizeof(float*) * num_matrices, hipMemcpyHostToDevice);

    // Get the flattened array
    float* gpu_array_buffer = mat_map[array_id];

    // Kernel launch parameters
    dim3 block_dim(256, 1, 1);
    dim3 grid_dim((arr_size / block_dim.x) + 1, 1, 1);

    // Run the kernels
    cuda_unflatten_array_kernel<<<grid_dim, block_dim>>>(gpu_array_buffer, arr_size, mat_rows, mat_cols, gpu_mat_buffers_ptr);
    gpuErrchk(hipPeekAtLastError());
}

__global__ void cuda_unflatten_array_strided_kernel(float* array_buffer, int arr_size, int num_matrices, int mat_rows, int mat_cols, float** mat_buffers) {
    const int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    const int array_buffer_index = tidX;

    if (array_buffer_index < arr_size) {
        // Check which mat_buffer to write to
        const int mat_buffer_index = array_buffer_index % num_matrices;
        const int mat_buffer_pixel = array_buffer_index / num_matrices;

        // Write result
        mat_buffers[mat_buffer_index][mat_buffer_pixel] = array_buffer[array_buffer_index];
    }
}

// Take an array and unflatten it into n same_dimension matrices. Each array's first n elements are the first elements in memory. [arr1_elem1, arr2_elem1, arr3_elem1, arr1_elem2, arr2_elem2, arr3_elem2, ...]
void cuda_unflatten_array_strided(size_t array_id, size_t arr_size, size_t mat_rows, size_t mat_cols, size_t* mat_ids) {
    int mat_size = mat_rows * mat_cols;
    int num_matrices = arr_size / mat_size;

    // Create the buffers for the matrices
    std::vector<float*> gpu_mat_buffers;
    for (int i = 0; i < num_matrices; i++) {
        size_t mat_id = register_matrix(mat_rows, mat_cols);
        gpu_mat_buffers.push_back(mat_map[mat_id]);

        // Write back to rust vector
        mat_ids[i] = mat_id;
    }

    // Upload the gpu_mat_buffers to the gpu
    float** gpu_mat_buffers_ptr;
    hipMallocAsync(&gpu_mat_buffers_ptr, sizeof(float*) * num_matrices, 0);
    hipMemcpy(gpu_mat_buffers_ptr, &gpu_mat_buffers[0], sizeof(float*) * num_matrices, hipMemcpyHostToDevice);

    // Get the flattened array
    float* gpu_array_buffer = mat_map[array_id];

    // Kernel launch parameters
    dim3 block_dim(256, 1, 1);
    dim3 grid_dim((arr_size / block_dim.x) + 1, 1, 1);

    // Run the kernels
    cuda_unflatten_array_strided_kernel<<<grid_dim, block_dim>>>(gpu_array_buffer, arr_size, num_matrices, mat_rows, mat_cols, gpu_mat_buffers_ptr);
    gpuErrchk(hipPeekAtLastError());
}