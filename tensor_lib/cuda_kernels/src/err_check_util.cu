#ifndef GPU_ERR_CHECK_UTIL
#define GPU_ERR_CHECK_UTIL

// Error checking macro: https://stackoverflow.com/a/14038590
#include <stdio.h>
#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

#endif