#include "hip/hip_runtime.h"
#include "./cuda_kernels.cuh"

__global__ void test_kernel() {
    printf("Hello from the kernel!\n");
}

#include <stdio.h>

#include "./cuda_kernels.cuh"

__global__ void add_kernel(int* result) {
    printf("dsijngkjdsg");
    *result = 8;
}

void test() {
    int result;
    int* d_result;

    hipError_t err = hipMalloc((void**)&d_result, sizeof(int));
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }

    add_kernel<<<1, 1>>>(d_result);
    if (hipPeekAtLastError() != hipSuccess) {
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
    }

    err = hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    hipFree(d_result);

    printf("Result: %d\n", result);
    std::cout << "Finished Running Kernels." << std::endl;
}
