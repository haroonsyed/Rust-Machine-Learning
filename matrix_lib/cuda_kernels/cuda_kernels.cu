#include "hip/hip_runtime.h"
#include "./cuda_kernels.cuh"

/////////////////////
/// TEST FUNCTIONS
/////////////////////
__global__ void test_kernel() {
    printf("Hello from the kernel!\n");
}

__global__ void test_kernel_2(int* result) {
    *result = 8;
}

void test() {
    int result;
    int* d_result;
    test_kernel<<<1, 1>>>();
    if (hipPeekAtLastError() != hipSuccess) {
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
    }

    hipError_t err = hipMalloc((void**)&d_result, sizeof(int));
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }

    test_kernel_2<<<1, 1>>>(d_result);
    if (hipPeekAtLastError() != hipSuccess) {
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
    }

    err = hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    hipFree(d_result);

    printf("Result: %d\n", result);
    std::cout << "Finished Running Kernels." << std::endl;
}

void test_array_fill(double* buffer, size_t length) {
    for (int i = 0; i < length; i++) {
        buffer[i] = i;
    }
}

/////////////////////
/// Matrix Setup API
/////////////////////
size_t register_matrix_buffer(double* gpu_buffer) {
    // Register with the map for retrieval later
    mat_map[mat_generated_count] = gpu_buffer;
    return mat_generated_count++;  // Fine if this overflows
}

size_t register_matrix(size_t rows, size_t cols) {
    // Upload the data
    double* gpu_buffer;
    hipMalloc(&gpu_buffer, sizeof(double) * rows * cols);

    return register_matrix_buffer(gpu_buffer);
}

size_t register_matrix(double* data, size_t rows, size_t cols) {
    // Upload the data
    double* gpu_buffer;
    hipMalloc(&gpu_buffer, sizeof(double) * rows * cols);
    hipMemcpy(gpu_buffer, data, sizeof(double) * rows * cols, hipMemcpyHostToDevice);

    return register_matrix_buffer(gpu_buffer);
}

void unregister_matrix(size_t mat_id) {
    hipFree(mat_map[mat_id]);
    mat_map.erase(mat_id);
}

void get_matrix_data(size_t mat_id, int rows, int cols, double* data_buffer) {
    double* gpu_buffer = mat_map[mat_id];
    hipMemcpy(data_buffer, gpu_buffer, sizeof(double) * rows * cols, hipMemcpyDeviceToHost);
}

//////////////////////////
/// Matrix Operations API
//////////////////////////
/// For now everything is naive implementations to pass tests
/// TODO: Optimize memory accesses for coalition (tidX is problematic since it accesses down rows. Try to have block handle data sequentially)
__global__ void element_add_kernel(double* mat1_buffer, int mat1_rows, int mat1_cols, double* mat2_buffer, int mat2_rows, int mat2_cols, double* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][j] + mat2[i][j]

        int index = tidX * out_cols + tidY;
        out_buffer[index] = mat1_buffer[index] + mat2_buffer[index];
    }
}

/// TODO: Abstract out the create buffer-> get gpu_buffer -> lanch kernel -> return result workflow
size_t cuda_element_add(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    double* gpu_mat1_buffer = mat_map[mat1_id];
    double* gpu_mat2_buffer = mat_map[mat2_id];
    double* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    element_add_kernel<<<block_dim, grid_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);

    // Return result matrix id
    return out_mat_id;
}

__global__ void element_subtract_kernel(double* mat1_buffer, int mat1_rows, int mat1_cols, double* mat2_buffer, int mat2_rows, int mat2_cols, double* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][j] - mat2[i][j]

        int index = tidX * out_cols + tidY;
        out_buffer[index] = mat1_buffer[index] - mat2_buffer[index];
    }
}

size_t cuda_element_subtract(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    double* gpu_mat1_buffer = mat_map[mat1_id];
    double* gpu_mat2_buffer = mat_map[mat2_id];
    double* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    element_subtract_kernel<<<block_dim, grid_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);

    // Return result matrix id
    return out_mat_id;
}

__global__ void element_multiply_kernel(double* mat1_buffer, int mat1_rows, int mat1_cols, double* mat2_buffer, int mat2_rows, int mat2_cols, double* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][j] * mat2[i][j]

        int index = tidX * out_cols + tidY;
        out_buffer[index] = mat1_buffer[index] * mat2_buffer[index];
    }
}

size_t cuda_element_multiply(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    double* gpu_mat1_buffer = mat_map[mat1_id];
    double* gpu_mat2_buffer = mat_map[mat2_id];
    double* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    element_multiply_kernel<<<block_dim, grid_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);

    // Return result matrix id
    return out_mat_id;
}

__global__ void scalar_multiply_kernel(double* mat1_buffer, int mat1_rows, int mat1_cols, double scalar, double* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][j] * scalar

        int index = tidX * out_cols + tidY;
        out_buffer[index] = mat1_buffer[index] * scalar;
    }
}

size_t cuda_scalar_multiply(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, double scalar) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    double* gpu_mat1_buffer = mat_map[mat1_id];
    double* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    scalar_multiply_kernel<<<block_dim, grid_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, scalar, gpu_out_buffer, out_rows, out_cols);

    // Return result matrix id
    return out_mat_id;
}

__global__ void matrix_multiply_kernel(double* mat1_buffer, int mat1_rows, int mat1_cols, double* mat2_buffer, int mat2_rows, int mat2_cols, double* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][:] weighted sum mat2[:][j]
        // Where common dimension : is mat1col/mat2row

        double weighted_sum = 0.0;
        for (int common = 0; common < mat1_cols; common++) {
            // mat1[i][common]
            int mat1_index = mat1_cols * tidX + common;
            // mat1[common][j]
            int mat2_index = mat2_cols * common + tidY;

            weighted_sum += mat1_buffer[mat1_index] * mat2_buffer[mat2_index];
        }

        int output_index = tidX * out_cols + tidY;
        out_buffer[output_index] = weighted_sum;
    }
}

size_t cuda_matrix_multiply(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat2_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    double* gpu_mat1_buffer = mat_map[mat1_id];
    double* gpu_mat2_buffer = mat_map[mat2_id];
    double* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    matrix_multiply_kernel<<<block_dim, grid_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);

    // Return result matrix id
    return out_mat_id;
}

__global__ void add_vector_to_columns_kernel(double* mat1_buffer, int mat1_rows, int mat1_cols, double* mat2_buffer, int mat2_rows, int mat2_cols, double* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][j] + mat2[i][0]

        int mat1_index = tidX * mat1_cols + tidY;
        int mat2_index = tidX;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] + mat2_buffer[mat2_index];
    }
}

size_t cuda_add_vector_to_columns(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    double* gpu_mat1_buffer = mat_map[mat1_id];
    double* gpu_mat2_buffer = mat_map[mat2_id];
    double* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    add_vector_to_columns_kernel<<<block_dim, grid_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);

    // Return result matrix id
    return out_mat_id;
}

__global__ void transpose_kernel(double* mat1_buffer, int mat1_rows, int mat1_cols, double* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[j][i]

        int mat1_index = tidY * mat1_cols + tidX;

        int output_index = tidX * out_cols + tidY;
        out_buffer[output_index] = mat1_buffer[mat1_index];
    }
}

size_t cuda_transpose(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = mat1_cols;
    int out_cols = mat1_rows;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    double* gpu_mat1_buffer = mat_map[mat1_id];
    double* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    transpose_kernel<<<block_dim, grid_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);

    // Return result matrix id
    return out_mat_id;
}

__global__ void sum_rows_kernel(double* mat1_buffer, int mat1_rows, int mat1_cols, double* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][0] = sum (mat1[i][:])

        double row_sum = 0.0;
        int mat1_row_start_index = tidX * mat1_cols;
        for (int i = 0; i < mat1_cols; i++) {
            int mat1_index = mat1_row_start_index + i;
            row_sum += mat1_buffer[mat1_index];
        }

        int output_index = tidX * out_cols + tidY;
        out_buffer[output_index] = row_sum;
    }
}

size_t cuda_sum_rows(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = 1;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    double* gpu_mat1_buffer = mat_map[mat1_id];
    double* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    sum_rows_kernel<<<block_dim, grid_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);

    // Return result matrix id
    return out_mat_id;
}

__global__ void sum_columns_kernel(double* mat1_buffer, int mat1_rows, int mat1_cols, double* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[0][j] = sum (mat1[:][j])

        double col_sum = 0.0;
        for (int i = 0; i < mat1_rows; i++) {
            int mat1_index = tidY + i * mat1_cols;
            col_sum += mat1_buffer[mat1_index];
        }

        int output_index = tidX * out_cols + tidY;
        out_buffer[output_index] = col_sum;
    }
}

size_t cuda_sum_columns(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = 1;
    int out_cols = mat1_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    double* gpu_mat1_buffer = mat_map[mat1_id];
    double* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    sum_columns_kernel<<<block_dim, grid_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);

    // Return result matrix id
    return out_mat_id;
}
