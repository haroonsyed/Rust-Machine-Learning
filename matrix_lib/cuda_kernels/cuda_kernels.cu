#include "hip/hip_runtime.h"
#include "./cuda_kernels.cuh"

__global__ void test_kernel() {
    printf("Hello from the kernel!\n");
}

#include <stdio.h>

#include "./cuda_kernels.cuh"

__global__ void test_kernel_2(int* result) {
    *result = 8;
}

void test() {
    int result;
    int* d_result;
    test_kernel<<<1, 1>>>();
    if (hipPeekAtLastError() != hipSuccess) {
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
    }

    hipError_t err = hipMalloc((void**)&d_result, sizeof(int));
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }

    test_kernel_2<<<1, 1>>>(d_result);
    if (hipPeekAtLastError() != hipSuccess) {
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
    }

    err = hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    hipFree(d_result);

    printf("Result: %d\n", result);
    std::cout << "Finished Running Kernels." << std::endl;
}
