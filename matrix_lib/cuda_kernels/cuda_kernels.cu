#include "hip/hip_runtime.h"
#include "./cuda_kernels.cuh"

__global__ void test_kernel() {
    printf("Hello from the kernel!\n");
}

#include <stdio.h>

#include "./cuda_kernels.cuh"

__global__ void add_kernel(int* result) {
    printf("dsijngkjdsg");
    *result = 8;
}

void test() {
    int result;
    int* d_result;

    hipMalloc((void**)&d_result, sizeof(int));

    add_kernel<<<1, 1>>>(d_result);

    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(d_result);

    printf("Result: %d\n", result);
    std::cout << "Finished Running Kernels." << std::endl;
}
