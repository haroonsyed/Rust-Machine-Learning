#include "hip/hip_runtime.h"
#include "./cuda_kernels.cuh"

__global__ void test_kernel() {
    printf("Hello from the kernel!\n");
}

__global__ void test_kernel_2(int* result) {
    *result = 8;
}

void test() {
    int result;
    int* d_result;
    test_kernel<<<1, 1>>>();
    if (hipPeekAtLastError() != hipSuccess) {
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
    }

    hipError_t err = hipMalloc((void**)&d_result, sizeof(int));
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }

    test_kernel_2<<<1, 1>>>(d_result);
    if (hipPeekAtLastError() != hipSuccess) {
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
    }

    err = hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    hipFree(d_result);

    printf("Result: %d\n", result);
    std::cout << "Finished Running Kernels." << std::endl;
}

void test_array_fill(double* buffer, size_t length) {
    for (int i = 0; i < length; i++) {
        buffer[i] = i;
    }
}

__global__ void matrix_multiply_kernel(double* mat1_buffer, int mat1_rows, int mat1_cols, double* mat2_buffer, int mat2_rows, int mat2_cols, double* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][:] weighted sum mat2[:][j]
        // Where common dimension : is mat1col/mat2row

        double weighted_sum = 0.0;
        for (int common = 0; common < mat1_cols; common++) {
            // mat1[i][common]
            int mat1_index = mat1_cols * tidX + common;
            // mat1[common][j]
            int mat2_index = mat2_cols * common + tidY;

            weighted_sum += mat1_buffer[mat1_index] * mat2_buffer[mat2_index];
        }

        int output_index = tidX * out_cols + tidY;
        out_buffer[output_index] = weighted_sum;
    }
}
void cuda_matrix_multiply(double* mat1_buffer, size_t mat1_rows, size_t mat1_cols, double* mat2_buffer, size_t mat2_rows, size_t mat2_cols, double* out_buffer, size_t out_rows, size_t out_cols) {
    // Setup the cuda buffers
    double* gpu_mat1_buffer;
    double* gpu_mat2_buffer;
    double* gpu_out_buffer;
    hipMalloc(&gpu_mat1_buffer, sizeof(double) * mat1_rows * mat1_cols);
    hipMalloc(&gpu_mat2_buffer, sizeof(double) * mat2_rows * mat2_cols);
    hipMalloc(&gpu_out_buffer, sizeof(double) * out_rows * out_cols);

    // Upload input data
    hipMemcpy(gpu_mat1_buffer, mat1_buffer, sizeof(double) * mat1_rows * mat1_cols, hipMemcpyHostToDevice);
    hipMemcpy(gpu_mat2_buffer, mat2_buffer, sizeof(double) * mat2_rows * mat2_cols, hipMemcpyHostToDevice);

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    matrix_multiply_kernel<<<block_dim, grid_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);

    // Download results to output
    hipMemcpy(out_buffer, gpu_out_buffer, sizeof(double) * out_rows * out_cols, hipMemcpyDeviceToHost);

    // Free resources
    hipFree(gpu_mat1_buffer);
    hipFree(gpu_mat2_buffer);
    hipFree(gpu_out_buffer);
}