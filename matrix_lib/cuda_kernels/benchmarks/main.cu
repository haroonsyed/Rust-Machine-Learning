#include <chrono>
#include <vector>
using namespace std::chrono;
#include "../cuda_kernels.cuh"

int main() {
    // This is just for timing, assumes everything is correct.
    // The tests already cover correctness.
    int dim = 4096;
    std::vector<float> data;
    for (int i = 0; i < dim * dim; i++) {
        data.push_back(23.47);
    }

    // Register
    int mat1 = register_matrix(&data[0], dim, dim);
    int mat2 = register_matrix(&data[0], dim, dim);

    auto start_host = high_resolution_clock::now();

    hipEvent_t start;
    hipEvent_t end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    int num_iter = 100;
    for (int i = 0; i < num_iter; i++) {
        // Perform multiplication
        int result_id = cuda_matrix_multiply(mat1, dim, dim, mat2, dim, dim);
        cuda_synchronize();
        unregister_matrix(result_id);
    }
    hipEventRecord(end);
    hipEventSynchronize(start);
    hipEventSynchronize(end);

    float gpu_time = 0;
    hipEventElapsedTime(&gpu_time, start, end);

    auto end_host = high_resolution_clock::now();
    auto cpu_time = duration_cast<milliseconds>(end_host - start_host);

    std::cout << "Average gpu function time was: " << gpu_time / num_iter << " ms" << std::endl;
    std::cout << "Including overhead was: " << (float)cpu_time.count() / num_iter << " ms" << std::endl;

    // Okay something is wrong with the overhead on rust benchmark. Something taking 184.3 ms here is taking 1.3 seconds there.
    // Same functions on ffi being called...
}