#include "hip/hip_runtime.h"
#include <unordered_map>

#include "./cuda_kernels.cuh"

bool init_cublas = false;
bool init_pool = false;
hipblasHandle_t handle;
size_t mat_generated_count(0);
std::unordered_map<size_t, float*> mat_map;

// Error checking macro: https://stackoverflow.com/a/14038590
#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = false) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/////////////////////
/// TEST FUNCTIONS
/////////////////////
__global__ void test_kernel() {
    printf("Hello from the kernel!\n");
}

__global__ void test_kernel_2(int* result) {
    *result = 8;
}

void test() {
    int result;
    int* d_result;
    test_kernel<<<1, 1>>>();
    gpuErrchk(hipPeekAtLastError());
    hipDeviceSynchronize();
    if (hipPeekAtLastError() != hipSuccess) {
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
    }

    hipError_t err = hipMalloc((void**)&d_result, sizeof(int));
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }

    test_kernel_2<<<1, 1>>>(d_result);
    if (hipPeekAtLastError() != hipSuccess) {
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
    }

    err = hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    hipFree(d_result);

    printf("Result: %d\n", result);
    printf("Finished Running Kernels.");
}

void test_array_fill(float* buffer, size_t length) {
    for (size_t i = 0; i < length; i++) {
        buffer[i] = i;
    }
}

void cuda_synchronize() {
    hipDeviceSynchronize();
}

/////////////////////
/// Matrix Setup API
/////////////////////
void init_cublas_handle() {
    hipblasCreate(&handle);
    hipblasSetStream(handle, 0);
    init_cublas = true;
}
void init_min_pool_size() {
    int device;
    hipGetDevice(&device);
    hipMemPool_t mempool;
    hipDeviceGetDefaultMemPool(&mempool, device);
    size_t threshold = sizeof(float) * 2048 * 2048;  // Around 68 Mb reserved
    hipMemPoolSetAttribute(mempool, hipMemPoolAttrReleaseThreshold, &threshold);
    init_pool = false;
}
size_t register_matrix_buffer(float* gpu_buffer) {
    if (init_pool) {
        init_min_pool_size();
    }

    // Register with the map for retrieval later
    mat_map[mat_generated_count] = gpu_buffer;
    return mat_generated_count++;  // Fine if this overflows
}

size_t register_matrix(size_t rows, size_t cols) {
    // Upload the data
    float* gpu_buffer;
    gpuErrchk(hipMallocAsync(&gpu_buffer, sizeof(float) * rows * cols, 0));

    return register_matrix_buffer(gpu_buffer);
}

size_t register_matrix(float* data, size_t rows, size_t cols) {
    // Upload the data
    float* gpu_buffer;
    gpuErrchk(hipMallocAsync(&gpu_buffer, sizeof(float) * rows * cols, 0));
    gpuErrchk(hipMemcpy(gpu_buffer, data, sizeof(float) * rows * cols, hipMemcpyHostToDevice));
    // Potentially nasty bug by acting like you copied data when you havent finished if using cudaMemCpyAsync...
    return register_matrix_buffer(gpu_buffer);
}

void unregister_matrix(size_t mat_id) {
    gpuErrchk(hipFreeAsync(mat_map[mat_id], 0));
    mat_map.erase(mat_id);
}

void get_matrix_data(size_t mat_id, int rows, int cols, float* data_buffer) {
    float* gpu_buffer = mat_map[mat_id];
    gpuErrchk(hipMemcpy(data_buffer, gpu_buffer, sizeof(float) * rows * cols, hipMemcpyDeviceToHost));
}

//////////////////////////
/// Matrix Operations API
//////////////////////////
/// TODO: Possibly grid stride. Optimize transpose. Optimize matmult.
__global__ void element_add_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;
    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] + mat2[i][j]
        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] + mat2_buffer[index];
    }
}
size_t cuda_element_add(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Create output buffer`
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);
    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];
    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);
    // Run the kernels
    element_add_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());
    // Return result matrix id
    return out_mat_id;
}

__global__ void element_subtract_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;
    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] - mat2[i][j]

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] - mat2_buffer[index];
    }
}

size_t cuda_element_subtract(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    element_subtract_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void element_multiply_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] * mat2[i][j]

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] * mat2_buffer[index];
    }
}

size_t cuda_element_multiply(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    element_multiply_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void scalar_multiply_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float scalar, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] * scalar

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] * scalar;
    }
}

size_t cuda_scalar_multiply(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, float scalar, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    scalar_multiply_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, scalar, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void matrix_multiply_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][:] weighted sum mat2[:][j]
        // Where common dimension : is mat1col/mat2row

        float weighted_sum = 0.0;
        for (int common = 0; common < mat1_cols; common++) {
            // mat1[i][common]
            int mat1_index = mat1_cols * tidX + common;
            // mat1[common][j]
            int mat2_index = mat2_cols * common + tidY;

            weighted_sum += mat1_buffer[mat1_index] * mat2_buffer[mat2_index];
        }

        int output_index = tidX * out_cols + tidY;
        out_buffer[output_index] = weighted_sum;
    }
}

__global__ void matrix_multiply_kernel_2(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    // Go by col row instead of row col. Enabled memory coalescing
    const int col = blockDim.x * blockIdx.x + threadIdx.x;
    const int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (row >= out_rows || col >= out_cols) {
        return;
    }

    // O[i][j] = mat1[i][:] weighted sum mat2[:][j]
    // Where common dimension : is mat1col/mat2row

    float weighted_sum = 0.0;
    for (int common = 0; common < mat1_cols; common++) {
        // mat1[i][common]
        int mat1_index = mat1_cols * row + common;
        // mat1[common][j]
        int mat2_index = mat2_cols * common + col;

        weighted_sum += mat1_buffer[mat1_index] * mat2_buffer[mat2_index];
    }

    const int output_index = row * out_cols + col;
    out_buffer[output_index] = weighted_sum;
}

__global__ void matrix_multiply_kernel_3(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    const int block_dim = 32;
    const int block_area = block_dim * block_dim;

    // Block tiling with shared memory
    __shared__ float s_mat1[block_area];
    __shared__ float s_mat2[block_area];

    const int block_row = blockIdx.y;
    const int block_col = blockIdx.x;

    int mat1_block_pos = block_row * block_dim * mat1_cols;
    int mat2_block_pos = block_col * block_dim;
    int out_block_pos = block_row * block_dim * out_cols + block_col * block_dim;

    // So within our block we are gonna figure out this thread's position
    int thread_row = threadIdx.y;
    int thread_col = threadIdx.x;

    int out_row = block_row * block_dim + thread_row;
    int out_col = block_col * block_dim + thread_col;
    if (out_row >= out_rows || out_col >= out_cols) {
        return;
    }

    float weighted_sum = 0.0;
    int common_partial_block = mat1_cols % block_dim;
    int common_in_block = mat1_cols - common_partial_block;
    for (int k = 0; k < common_in_block; k += block_dim) {
        s_mat1[thread_row * block_dim + thread_col] = mat1_buffer[mat1_block_pos + thread_row * mat1_cols + thread_col];
        s_mat2[thread_row * block_dim + thread_col] = mat2_buffer[mat2_block_pos + thread_row * mat2_cols + thread_col];
        __syncthreads();

        mat1_block_pos += block_dim;
        mat2_block_pos += block_dim * mat2_cols;
        for (int i = 0; i < block_dim; i++) {
            weighted_sum += s_mat1[thread_row * block_dim + i] * s_mat2[i * block_dim + thread_col];
        }
        __syncthreads();
    }

    // Handle partial block case
    s_mat1[thread_row * block_dim + thread_col] = mat1_buffer[mat1_block_pos + thread_row * mat1_cols + thread_col];
    s_mat2[thread_row * block_dim + thread_col] = mat2_buffer[mat2_block_pos + thread_row * mat2_cols + thread_col];
    __syncthreads();

    mat1_block_pos += block_dim;
    mat2_block_pos += block_dim * mat2_cols;
    for (int i = 0; i < common_partial_block; i++) {
        weighted_sum += s_mat1[thread_row * block_dim + i] * s_mat2[i * block_dim + thread_col];
    }

    out_buffer[out_block_pos + (thread_row * out_cols) + thread_col] = weighted_sum;
}

// block_M is rows in mat1 shared block
// block_N is cols in mat2 shared block
// block_k is shared dimensions for shared block. Also the # of results each thread will compute in C
// For this to work we want the shared dimension block_K to be smaller than block_M and block_N
// This way, multiple threads reuse sections from mat1 and mat2 ,with more output work
// Example: bK is 8 while bM and bN are 64. Output is a 64x64 area.
//          So you can spin up 512 threads per block. They load vram->shared
//          Then each thread can work on 8 pieces of the output 64x64 area (64*64/8 = 512)
template <const int block_M, const int block_N, const int block_K>
__global__ void matrix_multiply_kernel_4(int M, int N, int K, float* mat1_buffer, float* mat2_buffer, float* out_buffer) {
    // Block tiling with shared memory
    // Each one of these threads will handle #block_K output result columns
    __shared__ float s_mat1[block_M * block_K];
    __shared__ float s_mat2[block_K * block_N];

    float thread_results[block_K] = {0.0};

    const int block_row = blockIdx.y;
    const int block_col = blockIdx.x;

    // Get starting positions of each block
    int mat1_block_pos = block_row * block_M * K;
    int mat2_block_pos = block_col * block_N;
    int out_block_pos = block_row * block_M * N + block_col * block_N;

    // Used to track if out of bounds
    const int mat1_load_index_row = block_row * block_M + threadIdx.x;
    const int mat2_load_index_col = block_col * block_N + threadIdx.x;
    int mat_common_index = threadIdx.y;
    const bool exceeded_mat1_row = mat1_load_index_row >= M;
    const bool exceeded_mat2_col = mat2_load_index_col >= N;

    // outer loop over block tiles
    for (unsigned int common_block = 0; common_block < K; common_block += block_K) {
        const int within_mat1 = (int)!(exceeded_mat1_row || mat_common_index >= K);
        const int within_mat2 = (int)!(mat_common_index >= K || exceeded_mat2_col);
        int mat1_load_index = mat1_block_pos + threadIdx.x * K + threadIdx.y;
        int mat2_load_index = mat2_block_pos + threadIdx.y * N + threadIdx.x;

        // Prevent loading OOB
        mat1_load_index *= within_mat1;
        mat2_load_index *= within_mat2;

        // Load block data into shared memory. Load 0 is OOB.
        s_mat1[threadIdx.x * block_K + threadIdx.y] = mat1_buffer[mat1_load_index] * within_mat1;
        s_mat2[threadIdx.y * block_N + threadIdx.x] = mat2_buffer[mat2_load_index] * within_mat2;
        __syncthreads();

        // Advance block
        mat1_block_pos += block_K;
        mat2_block_pos += block_K * N;
        mat_common_index += block_K;

        // Go through common dimensions of block (across row of mat1 and down col of mat2)
        for (unsigned int block_common_index = 0; block_common_index < block_K; ++block_common_index) {
            const float shared_mat2_val = s_mat2[block_common_index * block_N + threadIdx.x];

            // Now this thread will accumulate the result for each t_row in the t_col of C
            for (unsigned int result_index = 0; result_index < block_K; ++result_index) {
                thread_results[result_index] +=
                    s_mat1[(threadIdx.y * block_K + result_index) * block_K + block_common_index] * shared_mat2_val;
            }
        }
        __syncthreads();
    }

    // Write results with bounds checking
    const int out_index_row = block_row * block_M + threadIdx.y * block_K;
    const int out_index_col = block_col * block_N + threadIdx.x;

    for (int i = 0; i < block_K; i++) {
        if (out_index_row + i < M && out_index_col < N) {
            out_buffer[out_block_pos + (threadIdx.y * block_K + i) * N + threadIdx.x] = thread_results[i];
        }
    }
}

// block_M is rows in mat1 shared block
// block_N is cols in mat2 shared block
// block_k is shared dimensions for shared block.
// The thread will calculate block_k * block_k results (So now a 2d version of v3)
// For this to work we want the shared dimension block_K to be extremely smaller than block_M and block_N
// This way, multiple threads reuse sections from mat1 and mat2 ,with more output work
// Example: bK is 8 while bM and bN are 128. Output is a 128x128 area.
//          So you can spin up 256 threads per block. They load vram->shared
//          Then each thread can work on 8x8 pieces of the output 128x128 area (128x128/64 = 256)
//          You might be wondering why not 512 threads like previously?
//          Well that increases the mem requirements per block, reducing occupancy.
template <const int block_M, const int block_N, const int block_K>
__global__ void matrix_multiply_kernel_5(int M, int N, int K, float* __restrict__ mat1_buffer, float* __restrict__ mat2_buffer, float* __restrict__ out_buffer) {
    // 2D Block tiling with shared memory
    __shared__ float s_mat1[block_M * block_K];
    __shared__ float s_mat2[block_K * block_N];

    float thread_results[block_K * block_K] = {0.0};

    const int block_row = blockIdx.y;
    const int block_col = blockIdx.x;

    // Output within block details
    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int out_block_row = tid / (block_M / block_K);
    const int out_block_col = tid % (block_N / block_K);

    const int num_threads_per_block = blockDim.x * blockDim.y;
    const int num_elements_to_load = (block_M * block_K) / num_threads_per_block;

    const int stride_mat1 = num_threads_per_block / block_K;
    const int stride_mat2 = num_threads_per_block / block_N;

    int mat1_pos = block_row * block_M * K;
    int mat2_pos = block_col * block_N;

// outer loop over block tiles
#pragma unroll
    for (int common_block = 0; common_block < K; common_block += block_K) {
#pragma unroll 4
        for (int i = 0; i < num_elements_to_load; i++) {
            const int mat1_row_within_block = (threadIdx.x + stride_mat1 * i);
            const int mat1_col_within_block = threadIdx.y;
            const int mat2_row_within_block = (threadIdx.y / num_elements_to_load) + i * stride_mat2;
            const int mat2_col_within_block = (threadIdx.y % num_elements_to_load) * blockDim.x + threadIdx.x;

            const int mat1_load_index_row = block_row * block_M + mat1_row_within_block;
            const int mat1_load_index_col = common_block + mat1_col_within_block;
            const int mat2_load_index_row = common_block + mat2_row_within_block;
            const int mat2_load_index_col = block_col * block_N + mat2_col_within_block;

            const bool exceeded_mat1_row = mat1_load_index_row >= M;
            const bool exceeded_mat1_col = mat1_load_index_col >= K;
            const bool exceeded_mat2_row = mat2_load_index_row >= K;
            const bool exceeded_mat2_col = mat2_load_index_col >= N;

            const int within_mat1 = (int)!(exceeded_mat1_row || exceeded_mat1_col);
            const int within_mat2 = (int)!(exceeded_mat2_row || exceeded_mat2_col);
            int mat1_load_index = mat1_pos + mat1_row_within_block * K + mat1_col_within_block;
            int mat2_load_index = mat2_pos + mat2_row_within_block * N + mat2_col_within_block;

            mat1_load_index *= within_mat1;
            mat2_load_index *= within_mat2;

            s_mat1[mat1_row_within_block * block_K + mat1_col_within_block] =
                mat1_buffer[mat1_load_index] * within_mat1;
            s_mat2[mat2_row_within_block * block_N + mat2_col_within_block] =
                mat2_buffer[mat2_load_index] * within_mat2;
        }

        mat1_pos += block_K;
        mat2_pos += block_K * N;

        __syncthreads();

        // Go through common dimensions of block (across row of mat1 and down col of mat2)
#pragma unroll 8
        for (int block_common_index = 0; block_common_index < block_K; block_common_index++) {
            // Now this thread will accumulate the block_K x block_K results from shared memory
#pragma unroll 8
            for (int result_index_row = 0; result_index_row < block_K; result_index_row++) {
#pragma unroll 8
                for (int result_index_col = 0; result_index_col < block_K; result_index_col++) {
                    thread_results[result_index_row * block_K + result_index_col] +=
                        s_mat1[(out_block_row * block_K + result_index_row) * block_K + block_common_index] *
                        s_mat2[(block_common_index * block_N) + (out_block_col * block_K + result_index_col)];
                }
            }
        }
        __syncthreads();
    }

    // Write results with bounds checking
    const int out_index_row = block_row * block_M + out_block_row * block_K;
    const int out_index_col = block_col * block_N + out_block_col * block_K;

#pragma unroll 8
    for (int i = 0; i < block_K; i++) {
#pragma unroll 8
        for (int j = 0; j < block_K; j++) {
            if (out_index_row + i < M && out_index_col + j < N) {
                out_buffer[(out_index_row + i) * N + out_index_col + j] = thread_results[i * block_K + j];
            }
        }
    }
}

size_t cuda_matrix_multiply(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat2_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    // const int THREADS_PER_BLOCK_X = 32;
    // const int THREADS_PER_BLOCK_Y = 32;

    // dim3 block_dim(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1);
    // dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // // Run the kernels
    // matrix_multiply_kernel_3<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);

    // V4 launch
    const int M = mat1_rows;
    const int N = mat2_cols;
    const int K = mat1_cols;

    const int THREADS_PER_BLOCK_X = 32;
    const int THREADS_PER_BLOCK_Y = 8;

    dim3 block_dim(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
    dim3 grid_dim((N / 128) + 1, (M / 128) + 1, 1);
    matrix_multiply_kernel_5<128, 128, 8><<<grid_dim, block_dim>>>(M, N, K, gpu_mat1_buffer, gpu_mat2_buffer, gpu_out_buffer);

    // CUBLAS version (for comparison to mine)
    // if (!init_cublas) {
    //     init_cublas_handle();
    // }
    // float alpha = 1.0;
    // float beta = 0.0;
    // hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, mat2_cols, mat1_rows, mat1_cols, &alpha, gpu_mat2_buffer, mat2_cols, gpu_mat1_buffer, mat1_cols, &beta, gpu_out_buffer, mat2_cols);

    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void add_vector_to_columns_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] + mat2[i][0]

        int mat1_index = tidY * mat1_cols + tidX;
        int mat2_index = tidY;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] + mat2_buffer[mat2_index];
    }
}

__global__ void add_vector_to_rows_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] + mat2[0][j]

        int mat1_index = tidY * mat1_cols + tidX;
        int mat2_index = tidX;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] + mat2_buffer[mat2_index];
    }
}

size_t cuda_add_vector(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Determine orientation
    bool is_column_vector = (mat2_cols == 1 && mat2_rows == mat1_rows);

    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    if (is_column_vector) {
        add_vector_to_columns_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    } else {
        add_vector_to_rows_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    }
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void divide_by_column_vector_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] / mat2[i][0]

        int mat1_index = tidY * mat1_cols + tidX;
        int mat2_index = tidY;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] / mat2_buffer[mat2_index];
    }
}

__global__ void divide_by_row_vector_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[i][j] / mat2[0][j]

        int mat1_index = tidY * mat1_cols + tidX;
        int mat2_index = tidX;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] / mat2_buffer[mat2_index];
    }
}

size_t cuda_divide_by_vector(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Determine orientation
    bool is_column_vector = (mat2_cols == 1 && mat2_rows == mat1_rows);

    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    if (is_column_vector) {
        divide_by_column_vector_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    } else {
        divide_by_row_vector_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    }
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void element_exp_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = exp(mat1[i][j])

        int index = tidY * out_cols + tidX;
        out_buffer[index] = exp(mat1_buffer[index]);  // Also available __exp for fast
    }
}

size_t cuda_element_exp(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    element_exp_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void element_ReLU_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = x if x>0 else 0

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] > 0 ? mat1_buffer[index] : 0.0;
    }
}

size_t cuda_element_ReLU(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    element_ReLU_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void element_ReLU_prime_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = x if x>0 else 1

        int index = tidY * out_cols + tidX;
        out_buffer[index] = mat1_buffer[index] == 0.0 ? 0.0 : 1.0;
    }
}

size_t cuda_element_ReLU_prime(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    element_ReLU_prime_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void sum_rows_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][0] = sum (mat1[i][:])

        float row_sum = 0.0;
        int mat1_row_start_index = tidY * mat1_cols;
        for (int i = 0; i < mat1_cols; i++) {
            int mat1_index = mat1_row_start_index + i;
            row_sum += mat1_buffer[mat1_index];
        }

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = row_sum;
    }
}

size_t cuda_sum_rows(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = 1;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    sum_rows_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void sum_columns_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[0][j] = sum (mat1[:][j])

        float col_sum = 0.0;
        for (int i = 0; i < mat1_rows; i++) {
            int mat1_index = tidX + i * mat1_cols;
            col_sum += mat1_buffer[mat1_index];
        }

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = col_sum;
    }
}

size_t cuda_sum_columns(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = 1;
    int out_cols = mat1_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    sum_columns_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void transpose_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // O[i][j] = mat1[j][i]

        int mat1_index = tidX * mat1_cols + tidY;

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = mat1_buffer[mat1_index];
    }
}

size_t cuda_transpose(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = mat1_cols;
    int out_cols = mat1_rows;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    transpose_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void cuda_max_pool_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // For each 2x2 area pick the maximum value
        // We will mem coalesce by getting first two in row 1
        // Then next 2 in row2

        // Grab data w/t bounds check
        // TODO: Bounds check

        int block_start_row = tidY * 2;
        int block_start_col = tidX * 2;
        int block_start = block_start_row * mat1_cols + block_start_col;

        // bool block_00_oob = false;
        bool block_01_oob = (block_start_col + 1) >= mat1_cols;
        bool block_10_oob = (block_start_row + 1) >= mat1_rows;
        bool block_11_oob = block_01_oob || block_10_oob;

        float small_float = -1e30;  // Should probably use FLT_MIN but language server no like it

        // TODO: Use bit operations instead of ternary (it's faster idk why the compiler can't figure it out)
        float block_00 = mat1_buffer[block_start];
        float block_01 = block_01_oob ? small_float : mat1_buffer[block_start + 1];
        block_start += mat1_cols;
        float block_10 = block_10_oob ? small_float : mat1_buffer[block_start];
        float block_11 = block_11_oob ? small_float : mat1_buffer[block_start + 1];

        float result = max(max(block_00, block_01), max(block_10, block_11));

        int output_index = tidY * out_cols + tidX;
        out_buffer[output_index] = result;
    }
}

// 2x2 since other reduction sizes are not really used
size_t cuda_max_pool(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = mat1_rows / 2 + mat1_rows % 2;
    int out_cols = mat1_cols / 2 + mat1_cols % 2;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    cuda_max_pool_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void cuda_rotate_180_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_cols && tidY < out_rows) {
        // Rotating an array 180 means
        // x_output = length - x_current
        // y_output = height - y_current
        int x_out = mat1_cols - tidX - 1;
        int y_out = mat1_rows - tidY - 1;
        int input = mat1_buffer[tidY * mat1_cols + tidX];

        int output_index = y_out * out_cols + x_out;
        out_buffer[output_index] = input;
    }
}

size_t cuda_rotate_180(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    cuda_rotate_180_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}