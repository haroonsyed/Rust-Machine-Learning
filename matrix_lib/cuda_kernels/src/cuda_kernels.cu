#include "hip/hip_runtime.h"
#include <unordered_map>

#include "./cuda_kernels.cuh"

bool init_cublas = false;
bool init_pool = false;
hipblasHandle_t handle;
size_t mat_generated_count(0);
std::unordered_map<size_t, float*> mat_map;

// Error checking macro: https://stackoverflow.com/a/14038590
#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = false) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/////////////////////
/// TEST FUNCTIONS
/////////////////////
__global__ void test_kernel() {
    printf("Hello from the kernel!\n");
}

__global__ void test_kernel_2(int* result) {
    *result = 8;
}

void test() {
    int result;
    int* d_result;
    test_kernel<<<1, 1>>>();
    gpuErrchk(hipPeekAtLastError());
    hipDeviceSynchronize();
    if (hipPeekAtLastError() != hipSuccess) {
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
    }

    hipError_t err = hipMalloc((void**)&d_result, sizeof(int));
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }

    test_kernel_2<<<1, 1>>>(d_result);
    if (hipPeekAtLastError() != hipSuccess) {
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
    }

    err = hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("The error is %s", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    hipFree(d_result);

    printf("Result: %d\n", result);
    printf("Finished Running Kernels.");
}

void test_array_fill(float* buffer, size_t length) {
    for (int i = 0; i < length; i++) {
        buffer[i] = i;
    }
}

void cuda_synchronize() {
    hipDeviceSynchronize();
}

/////////////////////
/// Matrix Setup API
/////////////////////
void init_cublas_handle() {
    hipblasCreate(&handle);
    hipblasSetStream(handle, 0);
    init_cublas = true;
}
void init_min_pool_size() {
    int device;
    hipGetDevice(&device);
    hipMemPool_t mempool;
    hipDeviceGetDefaultMemPool(&mempool, device);
    size_t threshold = sizeof(float) * 2048 * 2048;  // Around 68 Mb reserved
    hipMemPoolSetAttribute(mempool, hipMemPoolAttrReleaseThreshold, &threshold);
    init_pool = false;
}
size_t register_matrix_buffer(float* gpu_buffer) {
    if (init_pool) {
        init_min_pool_size();
    }

    // Register with the map for retrieval later
    mat_map[mat_generated_count] = gpu_buffer;
    return mat_generated_count++;  // Fine if this overflows
}

size_t register_matrix(size_t rows, size_t cols) {
    // Upload the data
    float* gpu_buffer;
    gpuErrchk(hipMallocAsync(&gpu_buffer, sizeof(float) * rows * cols, 0));

    return register_matrix_buffer(gpu_buffer);
}

size_t register_matrix(float* data, size_t rows, size_t cols) {
    // Upload the data
    float* gpu_buffer;
    gpuErrchk(hipMallocAsync(&gpu_buffer, sizeof(float) * rows * cols, 0));
    gpuErrchk(hipMemcpy(gpu_buffer, data, sizeof(float) * rows * cols, hipMemcpyHostToDevice));
    // Potentially nasty bug by acting like you copied data when you havent finished if using cudaMemCpyAsync...
    return register_matrix_buffer(gpu_buffer);
}

void unregister_matrix(size_t mat_id) {
    gpuErrchk(hipFreeAsync(mat_map[mat_id], 0));
    mat_map.erase(mat_id);
}

void get_matrix_data(size_t mat_id, int rows, int cols, float* data_buffer) {
    float* gpu_buffer = mat_map[mat_id];
    gpuErrchk(hipMemcpy(data_buffer, gpu_buffer, sizeof(float) * rows * cols, hipMemcpyDeviceToHost));
}

//////////////////////////
/// Matrix Operations API
//////////////////////////
/// For now everything is naive implementations to pass tests
/// TODO: Optimize memory accesses for coalition (tidX is problematic since it accesses down rows. Try to have block handle data sequentially). Possibly grid stride too.
__global__ void element_add_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][j] + mat2[i][j]

        int index = tidX * out_cols + tidY;
        out_buffer[index] = mat1_buffer[index] + mat2_buffer[index];
    }
}

size_t cuda_element_add(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    element_add_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void element_subtract_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][j] - mat2[i][j]

        int index = tidX * out_cols + tidY;
        out_buffer[index] = mat1_buffer[index] - mat2_buffer[index];
    }
}

size_t cuda_element_subtract(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    element_subtract_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void element_multiply_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][j] * mat2[i][j]

        int index = tidX * out_cols + tidY;
        out_buffer[index] = mat1_buffer[index] * mat2_buffer[index];
    }
}

size_t cuda_element_multiply(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    element_multiply_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void scalar_multiply_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float scalar, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][j] * scalar

        int index = tidX * out_cols + tidY;
        out_buffer[index] = mat1_buffer[index] * scalar;
    }
}

size_t cuda_scalar_multiply(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, float scalar, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    scalar_multiply_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, scalar, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void matrix_multiply_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][:] weighted sum mat2[:][j]
        // Where common dimension : is mat1col/mat2row

        float weighted_sum = 0.0;
        for (int common = 0; common < mat1_cols; common++) {
            // mat1[i][common]
            int mat1_index = mat1_cols * tidX + common;
            // mat1[common][j]
            int mat2_index = mat2_cols * common + tidY;

            weighted_sum += mat1_buffer[mat1_index] * mat2_buffer[mat2_index];
        }

        int output_index = tidX * out_cols + tidY;
        out_buffer[output_index] = weighted_sum;
    }
}

__global__ void matrix_multiply_kernel_2(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    // Go by col row instead of row col. Enabled memory coalescing
    const int col = blockDim.x * blockIdx.x + threadIdx.x;
    const int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (row >= out_rows || col >= out_cols) {
        return;
    }

    // O[i][j] = mat1[i][:] weighted sum mat2[:][j]
    // Where common dimension : is mat1col/mat2row

    float weighted_sum = 0.0;
    for (int common = 0; common < mat1_cols; common++) {
        // mat1[i][common]
        int mat1_index = mat1_cols * row + common;
        // mat1[common][j]
        int mat2_index = mat2_cols * common + col;

        weighted_sum += mat1_buffer[mat1_index] * mat2_buffer[mat2_index];
    }

    const int output_index = row * out_cols + col;
    out_buffer[output_index] = weighted_sum;
}

__global__ void matrix_multiply_kernel_3(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    const int block_dim = 32;
    const int block_area = block_dim * block_dim;

    // Block tiling with shared memory
    __shared__ float s_mat1[block_area];
    __shared__ float s_mat2[block_area];

    const int block_row = blockIdx.y;
    const int block_col = blockIdx.x;

    int mat1_block_pos = block_row * block_dim * mat1_cols;
    int mat2_block_pos = block_col * block_dim;
    int out_block_pos = block_row * block_dim * out_cols + block_col * block_dim;

    // So within our block we are gonna figure out this thread's position
    // int thread_row = threadIdx.x / block_dim;
    // int thread_col = threadIdx.x % block_dim;
    int thread_row = threadIdx.y;
    int thread_col = threadIdx.x;

    int out_row = block_row * block_dim + thread_row;
    int out_col = block_col * block_dim + thread_col;
    if (out_row >= out_rows || out_col >= out_cols) {
        return;
    }

    float weighted_sum = 0.0;
    int common_partial_block = mat1_cols % block_dim;
    int common_in_block = mat1_cols - common_partial_block;
    for (int k = 0; k < common_in_block; k += block_dim) {
        s_mat1[thread_row * block_dim + thread_col] = mat1_buffer[mat1_block_pos + thread_row * mat1_cols + thread_col];
        s_mat2[thread_row * block_dim + thread_col] = mat2_buffer[mat2_block_pos + thread_row * mat2_cols + thread_col];
        __syncthreads();

        mat1_block_pos += block_dim;
        mat2_block_pos += block_dim * mat2_cols;
        for (int i = 0; i < block_dim; i++) {
            weighted_sum += s_mat1[thread_row * block_dim + i] * s_mat2[i * block_dim + thread_col];
        }
        __syncthreads();
    }

    // Handle partial block case
    s_mat1[thread_row * block_dim + thread_col] = mat1_buffer[mat1_block_pos + thread_row * mat1_cols + thread_col];
    s_mat2[thread_row * block_dim + thread_col] = mat2_buffer[mat2_block_pos + thread_row * mat2_cols + thread_col];
    __syncthreads();

    mat1_block_pos += block_dim;
    mat2_block_pos += block_dim * mat2_cols;
    for (int i = 0; i < common_partial_block; i++) {
        weighted_sum += s_mat1[thread_row * block_dim + i] * s_mat2[i * block_dim + thread_col];
    }

    out_buffer[out_block_pos + (thread_row * out_cols) + thread_col] = weighted_sum;
}

size_t cuda_matrix_multiply(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat2_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK_X = 32;
    const int THREADS_PER_BLOCK_Y = 32;

    dim3 block_dim(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1);
    dim3 grid_dim((out_cols / block_dim.x) + 1, (out_rows / block_dim.y) + 1, 1);

    // Run the kernels
    matrix_multiply_kernel_3<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);

    // CUBLAS version (for comparison to mine)
    // if (!init_cublas) {
    //     init_cublas_handle();
    // }
    // float alpha = 1.0;
    // float beta = 0.0;
    // hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, mat2_cols, mat1_rows, mat1_cols, &alpha, gpu_mat2_buffer, mat2_cols, gpu_mat1_buffer, mat1_cols, &beta, gpu_out_buffer, mat2_cols);

    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void add_vector_to_columns_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][j] + mat2[i][0]

        int mat1_index = tidX * mat1_cols + tidY;
        int mat2_index = tidX;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] + mat2_buffer[mat2_index];
    }
}

__global__ void add_vector_to_rows_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][j] + mat2[0][j]

        int mat1_index = tidX * mat1_cols + tidY;
        int mat2_index = tidY;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] + mat2_buffer[mat2_index];
    }
}

size_t cuda_add_vector(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Determine orientation
    bool is_column_vector = (mat2_cols == 1 && mat2_rows == mat1_rows);

    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    if (is_column_vector) {
        add_vector_to_columns_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    } else {
        add_vector_to_rows_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    }
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void divide_by_column_vector_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][j] / mat2[i][0]

        int mat1_index = tidX * mat1_cols + tidY;
        int mat2_index = tidX;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] / mat2_buffer[mat2_index];
    }
}

__global__ void divide_by_row_vector_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* mat2_buffer, int mat2_rows, int mat2_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[i][j] / mat2[0][j]

        int mat1_index = tidX * mat1_cols + tidY;
        int mat2_index = tidY;

        int output_index = mat1_index;
        out_buffer[output_index] = mat1_buffer[mat1_index] / mat2_buffer[mat2_index];
    }
}

size_t cuda_divide_by_vector(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, size_t mat2_id, size_t mat2_rows, size_t mat2_cols, bool inplace) {
    // Determine orientation
    bool is_column_vector = (mat2_cols == 1 && mat2_rows == mat1_rows);

    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_mat2_buffer = mat_map[mat2_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    if (is_column_vector) {
        divide_by_column_vector_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    } else {
        divide_by_row_vector_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_mat2_buffer, mat2_rows, mat2_cols, gpu_out_buffer, out_rows, out_cols);
    }
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void element_exp_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = exp(mat1[i][j])

        int index = tidX * out_cols + tidY;
        out_buffer[index] = exp(mat1_buffer[index]);  // Also available __exp for fast
    }
}

size_t cuda_element_exp(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    element_exp_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void element_ReLU_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = x if x>0 else 0

        int index = tidX * out_cols + tidY;
        out_buffer[index] = mat1_buffer[index] > 0 ? mat1_buffer[index] : 0.0;
    }
}

size_t cuda_element_ReLU(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    element_ReLU_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void element_ReLU_prime_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = x if x>0 else 1

        int index = tidX * out_cols + tidY;
        out_buffer[index] = mat1_buffer[index] == 0.0 ? 0.0 : 1.0;
    }
}

size_t cuda_element_ReLU_prime(size_t mat1_id, size_t mat1_rows, size_t mat1_cols, bool inplace) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = mat1_cols;
    size_t out_mat_id = inplace ? mat1_id : register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    element_ReLU_prime_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void sum_rows_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][0] = sum (mat1[i][:])

        float row_sum = 0.0;
        int mat1_row_start_index = tidX * mat1_cols;
        for (int i = 0; i < mat1_cols; i++) {
            int mat1_index = mat1_row_start_index + i;
            row_sum += mat1_buffer[mat1_index];
        }

        int output_index = tidX * out_cols + tidY;
        out_buffer[output_index] = row_sum;
    }
}

size_t cuda_sum_rows(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = mat1_rows;
    int out_cols = 1;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    sum_rows_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void sum_columns_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[0][j] = sum (mat1[:][j])

        float col_sum = 0.0;
        for (int i = 0; i < mat1_rows; i++) {
            int mat1_index = tidY + i * mat1_cols;
            col_sum += mat1_buffer[mat1_index];
        }

        int output_index = tidX * out_cols + tidY;
        out_buffer[output_index] = col_sum;
    }
}

size_t cuda_sum_columns(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = 1;
    int out_cols = mat1_cols;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    sum_columns_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}

__global__ void transpose_kernel(float* mat1_buffer, int mat1_rows, int mat1_cols, float* out_buffer, int out_rows, int out_cols) {
    int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidX < out_rows && tidY < out_cols) {
        // O[i][j] = mat1[j][i]

        int mat1_index = tidY * mat1_cols + tidX;

        int output_index = tidX * out_cols + tidY;
        out_buffer[output_index] = mat1_buffer[mat1_index];
    }
}

size_t cuda_transpose(size_t mat1_id, size_t mat1_rows, size_t mat1_cols) {
    // Create output buffer
    int out_rows = mat1_cols;
    int out_cols = mat1_rows;
    size_t out_mat_id = register_matrix(out_rows, out_cols);

    // Get the gpu buffers to operate on
    float* gpu_mat1_buffer = mat_map[mat1_id];
    float* gpu_out_buffer = mat_map[out_mat_id];

    // Kernel launch parameters
    const int THREADS_PER_BLOCK = 32;
    dim3 block_dim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    dim3 grid_dim((out_rows / block_dim.x) + 1, (out_cols / block_dim.y) + 1, 1);

    // Run the kernels
    transpose_kernel<<<grid_dim, block_dim>>>(gpu_mat1_buffer, mat1_rows, mat1_cols, gpu_out_buffer, out_rows, out_cols);
    gpuErrchk(hipPeekAtLastError());

    // Return result matrix id
    return out_mat_id;
}